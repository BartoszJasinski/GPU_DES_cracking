
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <string>
#include <iostream>
#include <bitset>
#include <sstream>
#include <stdlib.h>

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}
//
//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    cudaError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // cudaDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = cudaDeviceReset();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//cudaError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    cudaError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = cudaSetDevice(0);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = cudaMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = cudaMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = cudaMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = cudaMemcpy(dev_a, a, size * sizeof(int), cudaMemcpyHostToDevice);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = cudaMemcpy(dev_b, b, size * sizeof(int), cudaMemcpyHostToDevice);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = cudaGetLastError();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // cudaDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = cudaDeviceSynchronize();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = cudaMemcpy(c, dev_c, size * sizeof(int), cudaMemcpyDeviceToHost);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    cudaFree(dev_c);
//    cudaFree(dev_a);
//    cudaFree(dev_b);
//    
//    return cudaStatus;
//}

////////////////////////////////////////////////////


int PC_1[56] = { 56, 48, 40, 32, 24, 16, 8, 0,
57, 49, 41, 33, 25, 17, 9, 1,
58, 50, 42, 34, 26, 18, 10, 2,
59, 51, 43, 35, 62, 54, 46, 38,
30, 22, 14, 6, 61, 53, 45, 37,
29, 21, 13, 5, 60, 52, 44, 36,
28, 20, 12, 4, 27, 19, 11, 3};


int shifts[] = { 1, 1, 2, 2, 2, 2, 2, 2, 1, 2, 2, 2, 2, 2, 2, 1 };

int PC_2[] = { 13, 16, 10, 23, 0, 4,
2, 27, 14, 5, 20, 9,
22, 18, 11, 3, 25, 7,
15, 6, 26, 19, 12, 1,
40, 51, 30, 36, 46, 54,
29, 39, 50, 44, 32, 47,
43, 48, 38, 55, 33, 52,
45, 41, 49, 35, 28, 31 };



void fun()
{
	for(int i = 0; i < 56; i++)
	{
		cout << PC_1[i] - 1 << ", ";

		if (!(i % 8))
			cout << "\n";
	}
}

template< typename T, size_t N, size_t M >
void printArray(T(&theArray)[N][M], int char_endl_nbr) {
	for (int x = 0; x < N; x++) 
	{
		for (int y = 0; y < M; y++)
		{
			cout << theArray[x][y];
			if (y == char_endl_nbr)
				cout << endl;
		}
		cout << endl;
	}
}


//template< typename T, size_t N, size_t M >
//void printArray2(T(&theArray)[N][M], int char_endl_nbr) {
//	for (int x = 0; x < N; x++)
//	{
//		for (int y = 0; y < M; y++)
//		{
//			cout << theArray[x][y];
//			if (y == char_endl_nbr)
//				cout << endl;
//			if (!(y % 6))
//				cout << " ";
//		}
//		cout << endl;
//	}
//}
template< typename T, size_t N, size_t M >
void printArray2(T(&theArray)[N][M], int char_endl_nbr) {
	for (int x = 0; x < N; x++)
	{
		for (int y = 0; y < M; y++)
		{
			cout << theArray[x][y];
			if (y == char_endl_nbr)
				cout << endl;
//			if (!(y % 6))
//				cout << " ";
		}
		cout << endl;
	}
}

//bitset bytesToBitset<int numBytes>(byte *data)
//{
//	std::bitset<numBytes * CHAR_BIT> b;
//
//	for (int i = 0; i < numBytes; ++i)
//	{
//		byte cur = data[i];
//		int offset = i * CHAR_BIT;
//
//		for (int bit = 0; bit < CHAR_BIT; ++bit)
//		{
//			b[offset] = cur & 1;
//			++offset;   // Move to next bit in b
//			cur >>= 1;  // Move to next bit in array
//		}
//	}
//
//	return b;
//}

//
//template<int numBytes>
//void bytesToBitset(string key_binary_ret)
//{
////	unsigned char c = 'a';
//
//	char const *c_key = key_binary_ret.c_str();
//
//	for (int i = 0; i < key_binary_ret.size(); i++)
//	{
//		for (int j = 0; j < 8; j++)
//		{
//
//			std::cout << ((c_key[i] >> j) & 1);
//		}
//		cout << " ";
//	}
//	
//}
//
//void bytesToBitset(string key_binary_ret)
//{
//	//	unsigned char c = 'a';
//
//	char const *c_key = key_binary_ret.c_str();
//
//	for (int i = 0; i < key_binary_ret.size(); i++)
//	{
//		for (int j = 0; j < 8; j++)
//		{
//
//			std::cout << ((c_key[i] >> j) & 1);
//		}
//		cout << " ";
//	}
//
//}
//
//int *get_bits(int n, int bitswanted) {
//	int *bits = (int *)malloc(sizeof(int) * bitswanted);
//
//	int k;
//	for (k = 0; k<bitswanted; k++) {
//		int mask = 1 << k;
//		int masked_n = n & mask;
//		int thebit = masked_n >> k;
//		bits[k] = thebit;
//	}
//
//	return bits;
//}
//

//template<int numBytes>
//bitset<numBytes * CHAR_BIT>bytesToBitset(char const *data)
//{
////	char const *data = key_binary_ret.c_str();
//	bitset<numBytes * CHAR_BIT> b = *data;
//
//	for (int i = 1; i < numBytes; ++i)
//	{
//		b <<= CHAR_BIT;  // Move to next bit in array
//		b |= data[i];    // Set the lowest CHAR_BIT bits
//	}
//
//	return b;
//}
//
//

void permutePC(int key_binary[], int key_binary_ret[], int key_binary_size, int PC[])
{
	for (int i = 0; i < key_binary_size; i++)
		key_binary_ret[i] = key_binary[PC[i]];

}

//C and D should have 28 array memebers
void createSubkeys(int key[], const int key_size, int C[], int D[], int CD_size, int run_number)
{
	const int size = key_size / 2;
	int tmp_C[28], tmp_D[28];
	for(int i = 0; i < key_size / 2; i++)
	{
		tmp_C[i] = key[i];
		tmp_D[i] = key[i + CD_size];
	}

	for(int i = 0; i < CD_size; i++)
	{
		C[i] = tmp_C[(i + shifts[run_number]) % CD_size];
		D[i] = tmp_D[(i + shifts[run_number]) % CD_size];
	}

}

void appendKeys(int leftKey[], int rightKey[], int key_size, int key_ret[])
{
	for(int i = 0; i < key_size; i++)
	{
		key_ret[i] = leftKey[i];
		key_ret[i + key_size] = rightKey[i];
	}
}

//key_binary_ret should be 64 bit long
string desEncyption(string message, int key_binary[], int key_size)
{
	int des_block_size_bytes = 8;
	int des_block_size_bits = 64;

//	cout << "omg";
	//DEBUG
//		cout << message.size();
//		cout << "\n" << message << "\n";

	if (message.size() % des_block_size_bytes)
		message.append(des_block_size_bytes - (message.size() % des_block_size_bytes), '0');//mayby another char to append  

	//DEBUG
//		cout << "\n" << message << "\n";
//		cout << message.size();

	int key_binary_ret[56];
	permutePC(key_binary, key_binary_ret, sizeof(key_binary_ret) / sizeof(key_binary_ret[0]), PC_1);

	//DEBUG
//	for (int i = 0; i < 56; i++)
//	{
//		if (!(i % 7))
//			cout << "\n";
//		cout << key_binary_ret[i];
//	}

	int subkeys_number = 17;
	int subkey_size = 28;
	int subkeys[17][56];
	int C[28], D[28];

	for (int i = 0; i < 56; i++)
		subkeys[0][i] = key_binary_ret[i];

	for(int i = 0; i < subkeys_number - 1; i++)
	{
		createSubkeys(subkeys[i], sizeof(key_binary_ret) / sizeof(key_binary_ret[0]), C, D, sizeof(C) / sizeof(C[0]), i);
		appendKeys(C, D, subkey_size, subkeys[i + 1]);
		//DEBUG

//		for(int i = 0; i < subkeys_number; i++)
//			for(int j = 0; j < 56; j++)
//				cout << 
//		for (int i = 0; i < 28; i++)
//		{
//			cout << C[i];
//		}
//		cout << endl;
//		for (int i = 0; i < 28; i++)
//		{
//			cout << D[i];
//		}
//		cout << endl;

	}

	//DEBUG
//	printArray(subkeys, 1000);

	int K[16][48];
	for(int i = 0; i < 16; i++)
	{
		permutePC(subkeys[i + 1], K[i], sizeof(K[0]) / sizeof(K[0][0]), PC_2);
	}

	//DEBUG
//	printArray2(K, 10000);

	return "NOT IMPLEMENTED";
}

//11110000110011001010101011110101010101100110011110001111
//11100001100110010101010111111010101011001100111100011110
//11000011001100101010101111110101010110011001111000111101
//00001100110010101010111111110101011001100111100011110101
//00110011001010101011111111000101100110011110001111010101
//11001100101010101111111100000110011001111000111101010101
//00110010101010111111110000111001100111100011110101010101
//11001010101011111111000011000110011110001111010101010110
//00101010101111111100001100111001111000111101010101011001
//01010101011111111000011001100011110001111010101010110011
//01010101111111100001100110011111000111101010101011001100
//01010111111110000110011001011100011110101010101100110011
//01011111111000011001100101010001111010101010110011001111
//01111111100001100110010101010111101010101011001100111100
//11111110000110011001010101011110101010101100110011110001
//11111000011001100101010101111010101010110011001111000111
//11110000110011001010101011110101010101100110011110001111

//000110110000001011101111111111000111000001110010
//011110011010111011011001110110111100100111100101
//010101011111110010001010010000101100111110011001
//011100101010110111010110110110110011010100011101
//011111001110110000000111111010110101001110101000
//011000111010010100111110010100000111101100101111
//111011001000010010110111111101100001100010111100
//111101111000101000111010110000010011101111111011
//111000001101101111101011111011011110011110000001
//101100011111001101000111101110100100011001001111
//001000010101111111010011110111101101001110000110
//011101010111000111110101100101000110011111101001
//100101111100010111010001111110101011101001000001
//010111110100001110110111111100101110011100111010
//101111111001000110001101001111010011111100001010
//110010110011110110001011000011100001011111110101



int main()
{
	string message = "0123456789ABCDEF", key = "133457799BBCDFF1";
	int key_binary[] = { 0,0,0,1,0,0,1,1, 0,0,1,1,0,1,0,0, 0,1,0,1,0,1,1,1, 0,1,1,1,1,0,0,1, 1,0,0,1,1,0,1,1, 1,0,1,1,1,1,0,0, 1,1,0,1,1,1,1,1, 1,1,1,1,0,0,0,1 };
	string cypherText = desEncyption(message, key_binary, sizeof(key_binary) / sizeof(key_binary[0]));


	//OLD
//	int key_hex = 0x133457799BBCDFF1;
///	int* bits = get_bits(key_hex, sizeof(key_hex) * CHAR_BIT);
///	
///	int cntr = 0;
///	while(bits[cntr])
///	{
///		cout << bits[cntr++];
///		if (!(cntr % 8))
///			cout << " ";
///	}
//	
//	//	bytesToBitset(key_hex);
//
////	stringstream ss;
///	ss << key_hex;
///	string test = "0";
///	bytesToBitset<16>(ss.str());
///	desEncyption(message, key_binary_ret);
//	
////	char const *c_key = key_binary_ret.c_str();
///	int c_key_size = 0;
///	while (c_key[c_key_size])
///	{
///		c_key_size++;
///	}
///
///	cout << c_key_size;
//
//
////	cout << CHAR_BIT;
//
//	cout << key_binary_ret.size();
//	bitset<17 * CHAR_BIT> bits = bytesToBitset<17>(key_binary_ret.c_str());
//
//	for (int i = 0; i < bits.count(); i++)
//	{
//		if (!(i % 8))
//			cout << " ";
//		cout << bits[i];
//	}
	//OLD




	return 0;

}



////////
////////
////////
//#include <string>
//#include <bitset>
//#include <type_traits>
//
//// SFINAE for safety. Sue me for putting it in a macro for brevity on the function
//#define IS_INTEGRAL(T) typename std::enable_if< std::is_integral<T>::value >::type* = 0
//
////template<class T>
////std::string integral_to_binary_string(T byte, IS_INTEGRAL(T))
////{
////	std::bitset<sizeof(T) * CHAR_BIT> bs(byte);
////	return bs.to_string();
////}
//
//template<class T>
//std::string integral_to_binary_string(T* byte, IS_INTEGRAL(T))
//{
//	std::bitset<sizeof(T) * CHAR_BIT> map[16];  // each bitset has all 64 bits set to 0
//
//	for (int i = 0; i < 8; i++)
//	{
//		std::bitset<sizeof(unsigned char) * CHAR_BIT> bs(byte[i]);
//		cout << bs.to_string();
//		if (!(i % 2))
//			cout << " ";
//	}
//	
//	return "";
//}
//
//int main() {
//	unsigned char byte = 0x133457799BBCDFF1; // 0000 0011
////	unsigned char byte_array[] = { 0x1, 0x3, 0x3, 0x4, 0x5, 0x7, 0x7, 0x9, 0x9, 0xB, 0xB, 0xC, 0xD, 0xF, 0xF, 0x1 };
//	unsigned char byte_array[] = { 0x13, 0x34, 0x57, 0x79, 0x9B, 0xBC, 0xDF, 0xF1};
//
//	std::cout << integral_to_binary_string(byte_array);
//	//std::cin.get();
//}