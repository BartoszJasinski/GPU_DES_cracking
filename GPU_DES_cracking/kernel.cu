#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "kernel.cuh"
#include "Utils.h"
#include "BinaryUtils.cuh"

#include <stdio.h>
#include <string>
#include <iostream>
#include <stdlib.h>
#include <vector>
#include <iomanip>

typedef unsigned char BYTE;

using namespace std;

int PC_1_size = 56, shifts_size = 16, PC_2_size = 48, IP_size = 64, E_size = 48, S_size_1 = 8, S_size_2 = 4, S_size_3 = 16, P_size = 32, IP_1_size = 64;

__device__ int key_cracked = 0;

const int PC_1[56] = { 49, 42, 35, 28, 21, 14, 7, 0,
50, 43, 36, 29, 22, 15, 8, 1,
51, 44, 37, 30, 23, 16, 9, 2,
52, 45, 38, 31, 55, 48, 41, 34,
27, 20, 13, 6, 54, 47, 40, 33,
26, 19, 12, 5, 53, 46, 39, 32,
25, 18, 11, 4, 24, 17, 10, 3 };


const int shifts[16] = { 1, 1, 2, 2, 2, 2, 2, 2, 1, 2, 2, 2, 2, 2, 2, 1 };

const int PC_2[48] = { 13, 16, 10, 23, 0, 4,
2, 27, 14, 5, 20, 9,
22, 18, 11, 3, 25, 7,
15, 6, 26, 19, 12, 1,
40, 51, 30, 36, 46, 54,
29, 39, 50, 44, 32, 47,
43, 48, 38, 55, 33, 52,
45, 41, 49, 35, 28, 31 };


const int IP[64] = { 57, 49, 41, 33, 25, 17, 9, 1,
59, 51, 43, 35, 27, 19, 11, 3,
61, 53, 45, 37, 29, 21, 13, 5,
63, 55, 47, 39, 31, 23, 15, 7,
56, 48, 40, 32, 24, 16,  8, 0,
58, 50, 42, 34, 26, 18, 10, 2,
60, 52, 44, 36, 28, 20, 12, 4,
62, 54, 46, 38, 30, 22, 14, 6 };

const int E[48] = { 31, 0, 1, 2, 3, 4,
3, 4, 5, 6, 7, 8,
7, 8, 9, 10, 11, 12,
11, 12, 13, 14, 15, 16,
15, 16, 17, 18, 19, 20,
19, 20, 21, 22, 23, 24,
23, 24, 25, 26, 27, 28,
27, 28, 29, 30, 31, 0 };

const int S[8][4][16] = { { { 14, 4, 13, 1, 2, 15, 11, 8, 3, 10, 6, 12, 5, 9, 0, 7 },
{ 0, 15, 7, 4, 14, 2, 13, 1, 10, 6, 12, 11, 9, 5, 3, 8 },
{ 4, 1, 14, 8, 13, 6, 2, 11, 15, 12, 9, 7, 3, 10, 5, 0 },
{ 15, 12, 8, 2, 4, 9, 1, 7, 5, 11, 3, 14, 10, 0, 6, 13 } },
{ { 15, 1, 8, 14, 6, 11, 3, 4, 9, 7, 2, 13, 12, 0, 5, 10 },
{ 3, 13, 4, 7, 15, 2, 8, 14, 12, 0, 1, 10, 6, 9, 11, 5 },
{ 0, 14, 7, 11, 10, 4, 13, 1, 5, 8, 12, 6, 9, 3, 2, 15 },
{ 13, 8, 10, 1, 3, 15, 4, 2, 11, 6, 7, 12, 0, 5, 14, 9 } },
{ { 10, 0, 9, 14, 6, 3, 15, 5, 1, 13, 12, 7, 11, 4, 2, 8 },
{ 13, 7, 0, 9, 3, 4, 6, 10, 2, 8, 5, 14, 12, 11, 15, 1 },
{ 13, 6, 4, 9, 8, 15, 3, 0, 11, 1, 2, 12, 5, 10, 14, 7 },
{ 1, 10, 13, 0, 6, 9, 8, 7, 4, 15, 14, 3, 11, 5, 2, 12 } },
{ { 7, 13, 14, 3, 0, 6, 9, 10, 1, 2, 8, 5, 11, 12, 4, 15 },
{ 13, 8, 11, 5, 6, 15, 0, 3, 4, 7, 2, 12, 1, 10, 14, 9 },
{ 10, 6, 9, 0, 12, 11, 7, 13, 15, 1, 3, 14, 5, 2, 8, 4 },
{ 3, 15, 0, 6, 10, 1, 13, 8, 9, 4, 5, 11, 12, 7, 2, 14 } },
{ { 2, 12, 4, 1, 7, 10, 11, 6, 8, 5, 3, 15, 13, 0, 14, 9 },
{ 14, 11, 2, 12, 4, 7, 13, 1, 5, 0, 15, 10, 3, 9, 8, 6 },
{ 4, 2, 1, 11, 10, 13, 7, 8, 15, 9, 12, 5, 6, 3, 0, 14 },
{ 11, 8, 12, 7, 1, 14, 2, 13, 6, 15, 0, 9, 10, 4, 5, 3 } },
{ { 12, 1, 10, 15, 9, 2, 6, 8, 0, 13, 3, 4, 14, 7, 5, 11 },
{ 10, 15, 4, 2, 7, 12, 9, 5, 6, 1, 13, 14, 0, 11, 3, 8 },
{ 9, 14, 15, 5, 2, 8, 12, 3, 7, 0, 4, 10, 1, 13, 11, 6 },
{ 4, 3, 2, 12, 9, 5, 15, 10, 11, 14, 1, 7, 6, 0, 8, 13 } },
{ { 4, 11, 2, 14, 15, 0, 8, 13, 3, 12, 9, 7, 5, 10, 6, 1 },
{ 13, 0, 11, 7, 4, 9, 1, 10, 14, 3, 5, 12, 2, 15, 8, 6 },
{ 1, 4, 11, 13, 12, 3, 7, 14, 10, 15, 6, 8, 0, 5, 9, 2 },
{ 6, 11, 13, 8, 1, 4, 10, 7, 9, 5, 0, 15, 14, 2, 3, 12 } },
{ { 13, 2, 8, 4, 6, 15, 11, 1, 10, 9, 3, 14, 5, 0, 12, 7 },
{ 1, 15, 13, 8, 10, 3, 7, 4, 12, 5, 6, 11, 0, 14, 9, 2 },
{ 7, 11, 4, 1, 9, 12, 14, 2, 0, 6, 10, 13, 15, 3, 5, 8 },
{ 2, 1, 14, 7, 4, 10, 8, 13, 15, 12, 9, 0, 3, 5, 6, 11 } }
};

const int P[] = { 15, 6, 19, 20,
28, 11, 27, 16,
0, 14, 22, 25,
4, 17, 30, 9,
1, 7, 23, 13,
31, 26, 2, 8,
18, 12, 29, 5,
21, 10, 3, 24 };

const int IP_1[] = {
	39, 7, 47, 15, 55, 23, 63, 31,
	38, 6, 46, 14, 54, 22, 62, 30,
	37, 5, 45, 13, 53, 21, 61, 29,
	36, 4, 44, 12, 52, 20, 60, 28,
	35, 3, 43, 11, 51, 19, 59, 27,
	34, 2, 42, 10, 50, 18, 58, 26,
	33, 1, 41, 9, 49, 17, 57, 25,
	32, 0, 40, 8, 48, 16, 56, 24 };


__constant__ int d_PC_1[56];

__constant__ int d_shifts[16];

__constant__ int d_PC_2[48];

__constant__ int d_IP[64];

__constant__ int d_E[48];

__constant__ int d_S[8][4][16];

__constant__ int d_P[32];

__constant__ int d_IP_1[64];




__device__ void permutePC(int key_binary[], int key_binary_ret[], int key_binary_size, const int PC[])
{
	for (int i = 0; i < key_binary_size; i++)
		key_binary_ret[i] = key_binary[PC[i]];

}

//C and D should have 28 array memebers
__device__ void createSubkeys(int key[], const int key_size, int C[], int D[], int CD_size, int run_number)
{
	const int size = key_size / 2;
	int tmp_C[28], tmp_D[28];
	for (int i = 0; i < key_size / 2; i++)
	{
		tmp_C[i] = key[i];
		tmp_D[i] = key[i + CD_size];
	}

	for (int i = 0; i < CD_size; i++)
	{
		C[i] = tmp_C[(i + d_shifts[run_number]) % CD_size];
		D[i] = tmp_D[(i + d_shifts[run_number]) % CD_size];
	}

}




__host__ __device__ void decimal2Binary(int decimal_int, int binary_int[], int run_number)
{
	if (decimal_int <= 1) {
		binary_int[run_number] = decimal_int;
		return;
	}

	int remainder = decimal_int % 2;
	decimal2Binary(decimal_int >> 1, binary_int, run_number + 1);
	binary_int[run_number] = remainder;
}



__device__ void reverseTab(int tab[], int tab_length)
{
	for (int i = 0; i < tab_length / 2; i++)
	{
		int tmp = tab[i];
		tab[i] = tab[tab_length - i - 1];
		tab[tab_length - i - 1] = tmp;

	}
}

__device__ void appendKeys(int leftKey[], int rightKey[], int key_size, int key_ret[])
{
	for (int i = 0; i < key_size; i++)
	{
		key_ret[i] = leftKey[i];
		key_ret[i + key_size] = rightKey[i];
	}
}


__device__ void expand(int R[], int tab_ret[], const int E[], int E_size)
{

	for (int i = 0; i < E_size; i++)
		tab_ret[i] = R[E[i]];
}

__device__ void xorArray(int first_tab[], int second_tab[], int tab_size, int tab_ret[])
{
	for (int i = 0; i < tab_size; i++)
		tab_ret[i] = (int)(!first_tab[i] != !second_tab[i]);

}

__device__ long long binary2Decimal(int binary_int[], int tab_length)
{
	long long dec = 0;

	for (int i = 0; i < tab_length; ++i)
	{
		int bin = binary_int[i];
		if (bin) dec = dec * 2 + 1;
		else dec *= 2;

	}

	return dec;
}

__device__ void f(int R[], int K[], int ret_tab[])
{
	int R_expanded[48];
	expand(R, R_expanded, d_E, 48);
	//DEBUG
	//	for (int i = 0; i < 48; i++)
	//	{
	//		if (!(i % 6))
	//			cout << " ";
	//		cout << R_expanded[i];
	//	}
	//	cout << endl << endl << endl;

	int xored[48];
	xorArray(K, R_expanded, 48, xored);
	//DEBUG
	//	for(int i = 0; i < 48; i++)
	//	{
	//		if (!(i % 6))
	//			cout << " ";
	//		cout << xored[i];
	//	}
	//	cout << endl << endl << endl;


	for (int i = 0; i < 8; i++)
	{
		int row[4] = { 0, 0, 0, 0 }, column[4] = { 0, 0, 0, 0 };
		row[3] = xored[6 * i + 5];
		row[2] = xored[6 * i];
		column[0] = xored[6 * i + 1];
		column[1] = xored[6 * i + 2];
		column[2] = xored[6 * i + 3];
		column[3] = xored[6 * i + 4];

		int chunk_length = 4;
		int R_chunk[4] = { 0, 0, 0, 0 };
		decimal2Binary(d_S[i][binary2Decimal(row, 4)][binary2Decimal(column, 4)], R_chunk, 0);
		reverseTab(R_chunk, chunk_length);
		for (int j = 0; j < chunk_length; j++)
		{
			R[4 * i + j] = R_chunk[j];
		}

	}
	//DEBUG
	//	for(int i = 0; i < 32; i++)
	//	{
	//		if (!(i % 4))
	//			cout << " ";
	//		cout << R[i];
	//	}
	//	cout << endl << endl << endl;

	permutePC(R, ret_tab, 32, d_P);
	//DEBUG
	//	for (int i = 0; i < 32; i++)
	//	{
	//		if (!(i % 4))
	//			cout << " ";
	//		cout << ret_tab[i];
	//	}
	//	cout << endl << endl << endl;

}


__device__ void reverse(int L[], int R[], int tab_length, int ret_tab[])
{
	for (int i = 0; i < tab_length; i++)
	{
		ret_tab[i] = R[i];
		ret_tab[i + tab_length] = L[i];
	}
}

__device__ void messageEncode(int message_binary[], int message_size, int K[][48], int msg_ret[])
{
	int L[32], R[32];
	for (int i = 0; i < message_size / 2; i++)
	{
		L[i] = message_binary[i];
		R[i] = message_binary[i + message_size / 2];
	}

	int prev_L[32], prev_R[32];
	for (int i = 0; i < message_size / 2; i++)
	{
		prev_L[i] = L[i];
		prev_R[i] = R[i];
	}

	for (int i = 0; i < 16; i++)
	{
		for (int j = 0; j < message_size / 2; j++)
			L[j] = prev_R[j];

		int tmp_f[32];
		f(prev_R, K[i], tmp_f);
		//DEBUG
		//		for (int i = 0; i < 32; i++)
		//		{
		//			if (!(i % 4))
		//				cout << " ";
		//			cout << tmp_f[i];
		//		}
		//		cout << endl << endl << endl;

		xorArray(prev_L, tmp_f, 32, R);

		//DEBUG
		//		for(int i = 0; i < 32; i++)
		//		{
		//			if (!(i % 4))
		//				cout << " ";
		//			cout << R[i];
		//		}
		//		cout << endl << endl << endl;

		//przepisanie R i L do prev_R i prev_L
		for (int j = 0; j < message_size / 2; j++)
		{
			prev_L[j] = L[j];
			prev_R[j] = R[j];
		}
	}

	int msg[64];
	reverse(L, R, 32, msg);
	//DEBUG
	//	for (int i = 0; i < 64; i++)
	//	{
	//		if (!(i % 8))
	//			cout << " ";
	//		cout << msg[i];
	//	}

	permutePC(msg, msg_ret, 64, d_IP_1);
	//DEBUG
	//	for(int i = 0; i < 64; i++)
	//	{
	//		if (!(i % 8))
	//			cout << " ";
	//		cout << msg_ret[i];
	//	}	

}





//key_binary_ret should be 64 bit long
__device__ void desEncryption(int message_binary[], int message_size, int key_binary[], int key_size, int msg_ret[])
{
	int des_block_size_bytes = 8;
	int des_block_size_bits = 64;

	//DEBUG
	//	printf("\n%s\n", "__device__ desEncryptionForDataBlock ");
	//	printf("%s\n", "message_binary");
	//	for (int i = 0; i < message_size; ++i)
	//	{
	//		printf("%i", message_binary[i]);
	//	}
	//		cout << message.size();
	//		cout << "\n" << message << "\n";
	//if (message.size() * CHAR_BIT != des_block_size_bits)
	//		cout << message.size() * CHAR_BIT;


	if (message_size % des_block_size_bytes)
	{
		//int tmp_message_binary[message_size + des_block_size_bytes - (message_size % des_block_size_bytes)]
		printf("%s\n", "KICIA");
		//	message_binary.append(des_block_size_bytes - (message.size() % des_block_size_bytes), '0');//mayby another char to append  
	}

	//OLD Verwsion with message as string 
	//	if (message.size() % des_block_size_bytes)
	//		message.append(des_block_size_bytes - (message.size() % des_block_size_bytes), '0');//mayby another char to append  

	//DEBUG
	//		cout << "\n" << message << "\n";
	//		cout << message.size();

	int key_binary_ret[56];
	permutePC(key_binary, key_binary_ret, sizeof(key_binary_ret) / sizeof(key_binary_ret[0]), d_PC_1);

	//DEBUG
	//	for (int i = 0; i < 56; i++)
	//	{
	//		if (!(i % 7))
	//			cout << "\n";
	//		cout << key_binary_ret[i];
	//	}

	int subkeys_number = 17;
	int subkey_size = 28;
	int subkeys[17][56];
	int C[28], D[28];

	for (int i = 0; i < 56; i++)
		subkeys[0][i] = key_binary_ret[i];

	for (int i = 0; i < subkeys_number - 1; i++)
	{
		createSubkeys(subkeys[i], sizeof(key_binary_ret) / sizeof(key_binary_ret[0]), C, D, sizeof(C) / sizeof(C[0]), i);
		appendKeys(C, D, subkey_size, subkeys[i + 1]);
		//DEBUG
		//		for(int i = 0; i < subkeys_number; i++)
		//			for(int j = 0; j < 56; j++)
		//				cout << 
		//		for (int i = 0; i < 28; i++)
		//		{
		//			cout << C[i];
		//		}
		//		cout << endl;
		//		for (int i = 0; i < 28; i++)
		//		{
		//			cout << D[i];
		//		}
		//		cout << endl;

	}

	//DEBUG
	//	printArray(subkeys, 1000);

	int K[16][48];
	for (int i = 0; i < 16; i++)
	{
		permutePC(subkeys[i + 1], K[i], sizeof(K[0]) / sizeof(K[0][0]), d_PC_2);
	}

	//DEBUG
	//	printArray2(K, 10000);

	//WARNING!!! message size 
	int message_binary_ret[64];
	permutePC(message_binary, message_binary_ret, message_size, d_IP);

	//DEBUG
	//	for(int i = 0; i < message_size; i++)
	//		cout << message_binary_ret[i];
	messageEncode(message_binary_ret, message_size, K, msg_ret);

}

const char* hexChar2Bin(char c)
{
	// TODO handle default / error
	switch (toupper(c))
	{
		case '0': return "0000";
		case '1': return "0001";
		case '2': return "0010";
		case '3': return "0011";
		case '4': return "0100";
		case '5': return "0101";
		case '6': return "0110";
		case '7': return "0111";
		case '8': return "1000";
		case '9': return "1001";
		case 'A': return "1010";
		case 'B': return "1011";
		case 'C': return "1100";
		case 'D': return "1101";
		case 'E': return "1110";
		case 'F': return "1111";
		default:
			return "ERROR_hexChar2Bin";
	}
}

std::string hex2Bin(const std::string& hex)
{
	// TODO use a loop from <algorithm> or smth
	std::string bin;
	for (unsigned i = 0; i != hex.length(); ++i)
		bin += hexChar2Bin(hex[i]);
	return bin;
}

void str2Int(string& str_int, int ret_int[], int ret_int_size)
{
	for (int i = 0; i < ret_int_size; i++)
		ret_int[i] = (str_int.c_str()[i] - '0');

}

string getHexStringFromBinaryString(string sHex)
{
	string sReturn = "";
	int bit_length = 4;
	const string const bins[] = { "0000", "0001", "0010", "0011", "0100", "0101", "0110", "0111",
		"1000", "1001", "1010", "1011", "1100", "1101", "1110", "1111" };
	for (int i = 0; i < sHex.length() / bit_length; ++i)
	{
		string s = sHex.substr(bit_length * i, bit_length);

		if (s == bins[0])
			sReturn.append("0");
		if (s == bins[1])
			sReturn.append("1");
		if (s == bins[2])
			sReturn.append("2");
		if (s == bins[3])
			sReturn.append("3");
		if (s == bins[4])
			sReturn.append("4");
		if (s == bins[5])
			sReturn.append("5");
		if (s == bins[6])
			sReturn.append("6");
		if (s == bins[7])
			sReturn.append("7");
		if (s == bins[8])
			sReturn.append("8");
		if (s == bins[9])
			sReturn.append("9");
		if (s == bins[10])
			sReturn.append("A");
		if (s == bins[11])
			sReturn.append("B");
		if (s == bins[12])
			sReturn.append("C");
		if (s == bins[13])
			sReturn.append("D");
		if (s == bins[14])
			sReturn.append("E");
		if (s == bins[15])
			sReturn.append("F");
	}

	return sReturn;

}

__host__ __device__ void consecutiveKeyGenerator(unsigned long long &present_key, int next_key_binary[], int next_key_binary_size)
{
	for (int i = 0; i < next_key_binary_size; i++)
		next_key_binary[i] = 0;
	decimal2Binary(present_key, next_key_binary, 0);
	//	present_key++;
}

__host__ __device__ bool compareArrays(int message[], int cyphertext[])
{
	for (int i = 0; i < 64; i++)
	{
		if (message[i] != cyphertext[i])
			return false;
	}

	return true;
}



__global__
void crackDes(int message_binary[], int cyphertext_binary[], int message_binary_size, unsigned long long computation_size)
{
	//printf("%s\n", "__global__ crackDes");



	//DEBUG
	//	for (int i = 0; i < possible_key_binary_size; ++i)
	//	{
	//		printf("%i", possible_key_binary[i]);
	//	}

	int msg_ret[64];

	//	printf("%s\n", "BEFORE desEncryptionForDataBlock");

	int possible_key_binary_size = 56;
	int possible_key_binary[56];
	unsigned long long present_key = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned long long last_key = present_key + computation_size;

	//unsigned long long temp = present_key + 2147483648;
	for (unsigned long long i = present_key; i < last_key; i++)
	{
		if (key_cracked == 1)
		{
			//		printf("%i", key_cracked);
			return;
		}
		consecutiveKeyGenerator(i, possible_key_binary, possible_key_binary_size);
		//	printf("%s\n", "BEFORE desEncryptionForDataBlock");
		desEncryption(message_binary, message_binary_size, possible_key_binary, 16, msg_ret);
		//	printf("%s\n", "AFTER desEncryptionForDataBlock");

		if (compareArrays(msg_ret, cyphertext_binary))
		{
			key_cracked = 1;
			printf("%s", "USED KEY IS: ");
			for (int i = 0; i < possible_key_binary_size; i++)
				printf("%i", possible_key_binary[i]);
			printf("\n");
		}
	}
}


__host__
void crackDes(string message, string cyphertext)
{
	string str_message = hex2Bin(message);
	int h_message_binary_size = 64;
	int h_message_binary[64];
	str2Int(str_message, h_message_binary, h_message_binary_size);

	string str_cyphertext = hex2Bin(cyphertext);
	int h_cyphertext_binary_size = 64;
	int h_cyphertext_binary[64];
	str2Int(str_cyphertext, h_cyphertext_binary, h_cyphertext_binary_size);

	int* d_message_binary = 0;
	hipMalloc((void**)&d_message_binary, h_message_binary_size * sizeof(int));
	hipMemcpy(d_message_binary, h_message_binary, h_message_binary_size * sizeof(int), hipMemcpyHostToDevice);

	int* d_cyphertext_binary = 0;
	hipMalloc((void**)&d_cyphertext_binary, h_cyphertext_binary_size * sizeof(int));
	hipMemcpy(d_cyphertext_binary, h_cyphertext_binary, h_cyphertext_binary_size * sizeof(int), hipMemcpyHostToDevice);

	const int threads_per_block = 512;//FERMI //1024; //2^10
	const int nbr_of_block_in_one_dim = 8192; //2 ^ 13;
	const int test_nbr_of_block = 32768; //2 ^ 15
	unsigned long long computation_size = pow(2, 47) / (nbr_of_block_in_one_dim);
	printf("%s\n", "__host__ crackDes BEFORE __device__ crackDes");
	crackDes<<<nbr_of_block_in_one_dim, threads_per_block>>>(d_message_binary, d_cyphertext_binary, h_message_binary_size, computation_size);
	printf("%s\n", "__host__ crackDes AFTER __device__ crackDes");
	//DEBUG
	//	for (int i = 0; i < 64; i++)
	//	{
	//		if (!(i % 8))
	//			cout << " ";
	//		cout << msg_ret[i];
	//	}

	//	string binary;
	//	for (int i = 0; i < 64; i++)
	//		binary.push_back(std::to_string(msg_ret[i]).c_str()[0]);
	//DEBUG
	//cout << binary;

}

__global__
void desEncryption(int message_binary[], int key_binary[], int message_binary_size, int msg_ret[])
{
	//DEBUG
	//	printf("%s\n", "before DEBUG __global__ desEncryptionForDataBlock MESSAGE_BINARY");
	//	for (int i = 0; i < message_binary_size; ++i)
	//	{
	//		printf("%i", message_binary[i]);
	//	}
	//	printf("%s\n", "after DEBUG __global__ desEncryptionForDataBlock MESSAGE_BINARY");

	//int msg_ret[64];
	//	printf("%s\n", "BEFORE desEncryptionForDataBlock");							14 should be here
	desEncryption(message_binary, message_binary_size, key_binary, 16, msg_ret);
	//	printf("%s\n", "before DEBUG __global__ desEncryptionForDataBlock MSG_RET");
	//	for (int i = 0; i < 64; ++i)
	//	{
	//		printf("%i", 123123123);
	//	}
	//	printf("%s\n", "after DEBUG __global__ desEncryptionForDataBlock MSG_RET");

}


__host__
string desEncryptionForDataBlock(string message, string key)
{
	string str_message = hex2Bin(message);
	int h_message_binary_size = 64;
	int h_message_binary[64];
	str2Int(str_message, h_message_binary, h_message_binary_size);

	string str_key = hex2Bin(key);
	int h_key_binary_size = 56;
	int h_key_binary[56];
	str2Int(str_key, h_key_binary, h_key_binary_size);

	int* d_message_binary = 0;
	hipMalloc((void**)&d_message_binary, h_message_binary_size * sizeof(int));
	hipMemcpy(d_message_binary, h_message_binary, h_message_binary_size * sizeof(int), hipMemcpyHostToDevice);

	int* d_key_binary = 0;
	hipMalloc((void**)&d_key_binary, h_key_binary_size * sizeof(int));
	hipMemcpy(d_key_binary, h_key_binary, h_key_binary_size * sizeof(int), hipMemcpyHostToDevice);

	int* d_msg_ret;
	hipMalloc((void**)&d_msg_ret, 64 * sizeof(int));

	//DEBUG
	//	printf("%s\n", "before DEBUG __host__ desEncryptionForDataBlock");
	//	for (int i = 0; i < 64; ++i)
	//	{
	//		printf("%i", h_message_binary[i]);
	//	}
	//	printf("%s\n", "after DEBUG __host__ desEncryptionForDataBlock");

	desEncryption<<<1, 1 >>>(d_message_binary, d_key_binary, 64, d_msg_ret);

	hipDeviceSynchronize();

	int* h_msg_ret = (int*)malloc(64 * sizeof(int));
	hipMemcpy(h_msg_ret, d_msg_ret, 64 * sizeof(int), hipMemcpyDeviceToHost);
	//DEBUG
	//	printf("\n%s\n", "before DEBUG __host__ desEncryptionForDataBlock H_MSG_RET");
	//	for (int i = 0; i < 64; ++i)
	//	{
	//		printf("%i", h_msg_ret[i]);
	//	}
	//	printf("%s\n", "after DEBUG __host__ desEncryptionForDataBlock H_MSG_RET");


	string binary;
	for (int i = 0; i < 64; i++)
		binary.push_back(std::to_string(h_msg_ret[i]).c_str()[0]);
	//DEBUG
	//cout << binary;

	return getHexStringFromBinaryString(binary);

}


__host__
string desEncryption(string message, string key)
{
	int block_size = 16;
	string encryptedMessage = "";
	for (int i = 0; i < message.size() / block_size; ++i)
		encryptedMessage += desEncryptionForDataBlock(message.substr(i * block_size, block_size), key);

	return encryptedMessage;
}


void initArrays()
{
	hipMemcpyToSymbol(HIP_SYMBOL(d_PC_1), PC_1, PC_1_size * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_shifts), shifts, shifts_size * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_PC_2), PC_2, PC_2_size * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_IP), IP, IP_size * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_E), E, E_size * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_P), P, P_size * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_S), S, S_size_1 * S_size_2 * S_size_3 * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_IP_1), IP_1, IP_1_size * sizeof(int));

}

void resizeGPUHeap()
{
	size_t size_heap, size_stack;
	hipDeviceSetLimit(hipLimitMallocHeapSize, 20000000 * sizeof(double));
	hipDeviceSetLimit(hipLimitStackSize, 12928);
	hipDeviceGetLimit(&size_heap, hipLimitMallocHeapSize);
	hipDeviceGetLimit(&size_stack, hipLimitStackSize);
	printf("Heap size found to be %d; Stack size found to be %d\n", (int)size_heap, (int)size_stack);

}





/*

bool inRange(int number, int lower_bound, int upper_bound)
{
	return (lower_bound <= number && number >= upper_bound);
}

void printArray(int array[], int size)
{
	for (int i = 0; i < size; ++i)
	{
		cout << array[i];
	}
	cout << "\n";
}

void tests()
{
	cout << sizeof(const int);

	/*unsigned long long last = 10;
	for (unsigned long long i = 0; i < last; i++)
	{
	int key_binary[56];
	consecutiveKeyGenerator(i, key_binary, 56);
	cout << "\t" << i << "\n";
	printArray(key_binary, 56);
	}


	cout << "PTYS" << endl;
	string key = "10000000000000";
	string str_key = hex2Bin(key);
	int h_key_binary_size = 56;
	int h_key_binary[56];
	str2Int(str_key, h_key_binary, h_key_binary_size);
	printArray(h_key_binary, h_key_binary_size);#1#
}
*/