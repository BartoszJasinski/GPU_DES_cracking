#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string>
#include <iostream>
#include <bitset>
#include <sstream>
#include <stdlib.h>
#include <vector>
#include <iomanip>

typedef unsigned char BYTE;

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}
//
//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}

////////////////////////////////////////////////////
//int S1[4][16] = { { 14, 4, 13, 1, 2, 15, 11, 8, 3, 10, 6, 12, 5, 9, 0, 7 },
//{ 0, 15, 7, 4, 14, 2, 13, 1, 10, 6, 12, 11, 9, 5, 3, 8 },
//{ 4, 1, 14, 8, 13, 6, 2, 11, 15, 12, 9, 7, 3, 10, 5, 0 },
//{ 15, 12, 8, 2, 4, 9, 1, 7, 5, 11, 3, 14, 10, 0, 6, 13 } };
//
//int S2[4][16] = { { 15, 1, 8, 14, 6, 11, 3, 4, 9, 7, 2, 13, 12, 0, 5, 10 },
//{ 3, 13, 4, 7, 15, 2, 8, 14, 12, 0, 1, 10, 6, 9, 11, 5 },
//{ 0, 14, 7, 11, 10, 4, 13, 1, 5, 8, 12, 6, 9, 3, 2, 15, },
//{ 13, 8, 10, 1, 3, 15, 4, 2, 11, 6, 7, 12, 0, 5, 14, 9 } };
//
//int S3[4][16] = { { 10, 0, 9, 14, 6, 3, 15, 5, 1, 13, 12, 7, 11, 4, 2, 8 },
//{ 13, 7, 0, 9, 3, 4, 6, 10, 2, 8, 5, 14, 12, 11, 15, 1 },
//{ 13, 6, 4, 9, 8, 15, 3, 0, 11, 1, 2, 12, 5, 10, 14, 7 },
//{ 1, 10, 13, 0, 6, 9, 8, 7, 4, 15, 14, 3, 11, 5, 2, 12 } };
//
//int S4[4][16] = { { 7, 13, 14, 3, 0, 6, 9, 10, 1, 2, 8, 5, 11, 12, 4, 15 },
//{ 13, 8, 11, 5, 6, 15, 0, 3, 4, 7, 2, 12, 1, 10, 14, 9 },
//{ 10, 6, 9, 0, 12, 11, 7, 13, 15, 1, 3, 14, 5, 2, 8, 4 },
//{ 3, 15, 0, 6, 10, 1, 13, 8, 9, 4, 5, 11, 12, 7, 2, 14 } };
//
//int S5[4][16] = { { 2, 12, 4, 1, 7, 10, 11, 6, 8, 5, 3, 15, 13, 0, 14, 9 },
//{ 14, 11, 2, 12, 4, 7, 13, 1, 5, 0, 15, 10, 3, 9, 8, 6 },
//{ 4, 2, 1, 11, 10, 13, 7, 8, 15, 9, 12, 5, 6, 3, 0, 14 },
//{ 11, 8, 12, 7, 1, 14, 2, 13, 6, 15, 0, 9, 10, 4, 5, 3 } };
//
//int S6[4][16] = { { 12, 1, 10, 15, 9, 2, 6, 8, 0, 13, 3, 4, 14, 7, 5, 11 },
//{ 10, 15, 4, 2, 7, 12, 9, 5, 6, 1, 13, 14, 0, 11, 3, 8 },
//{ 9, 14, 15, 5, 2, 8, 12, 3, 7, 0, 4, 10, 1, 13, 11, 6 },
//{ 4, 3, 2, 12, 9, 5, 15, 10, 11, 14, 1, 7, 6, 0, 8, 13 } };
//
//int S7[4][16] = { { 4, 11, 2, 14, 15, 0, 8, 13, 3, 12, 9, 7, 5, 10, 6, 1 },
//{ 13, 0, 11, 7, 4, 9, 1, 10, 14, 3, 5, 12, 2, 15, 8, 6 },
//{ 1, 4, 11, 13, 12, 3, 7, 14, 10, 15, 6, 8, 0, 5, 9, 2 },
//{ 6, 11, 13, 8, 1, 4, 10, 7, 9, 5, 0, 15, 14, 2, 3, 12 } };
//
//int S8[4][16] = { { 13, 2, 8, 4, 6, 15, 11, 1, 10, 9, 3, 14, 5, 0, 12, 7 },
//{ 1, 15, 13, 8, 10, 3, 7, 4, 12, 5, 6, 11, 0, 14, 9, 2 },
//{ 7, 11, 4, 1, 9, 12, 14, 2, 0, 6, 10, 13, 15, 3, 5, 8 },
//{ 2, 1, 14, 7, 4, 10, 8, 13, 15, 12, 9, 0, 3, 5, 6, 11 } };


int PC_1[56] = { 56, 48, 40, 32, 24, 16, 8, 0,
57, 49, 41, 33, 25, 17, 9, 1,
58, 50, 42, 34, 26, 18, 10, 2,
59, 51, 43, 35, 62, 54, 46, 38,
30, 22, 14, 6, 61, 53, 45, 37,
29, 21, 13, 5, 60, 52, 44, 36,
28, 20, 12, 4, 27, 19, 11, 3};


int shifts[] = { 1, 1, 2, 2, 2, 2, 2, 2, 1, 2, 2, 2, 2, 2, 2, 1 };

int PC_2[] = { 13, 16, 10, 23, 0, 4,
2, 27, 14, 5, 20, 9,
22, 18, 11, 3, 25, 7,
15, 6, 26, 19, 12, 1,
40, 51, 30, 36, 46, 54,
29, 39, 50, 44, 32, 47,
43, 48, 38, 55, 33, 52,
45, 41, 49, 35, 28, 31 };


int IP[] = { 57, 49, 41, 33, 25, 17, 9, 1,
59, 51, 43, 35, 27, 19, 11, 3,
61, 53, 45, 37, 29, 21, 13, 5,
63, 55, 47, 39, 31, 23, 15, 7,
56, 48, 40, 32, 24, 16,  8, 0,
58, 50, 42, 34, 26, 18, 10, 2,
60, 52, 44, 36, 28, 20, 12, 4,
62, 54, 46, 38, 30, 22, 14, 6 };

int E[] = { 31, 0, 1, 2, 3, 4,
3, 4, 5, 6, 7, 8,
7, 8, 9, 10, 11, 12,
11, 12, 13, 14, 15, 16,
15, 16, 17, 18, 19, 20,
19, 20, 21, 22, 23, 24,
23, 24, 25, 26, 27, 28,
27, 28, 29, 30, 31, 0 };
//sprawdzi� s box
int S[8][4][16] = { { {14, 4, 13, 1, 2, 15, 11, 8, 3, 10, 6, 12, 5, 9, 0, 7},
{0, 15, 7, 4, 14, 2, 13, 1, 10, 6, 12, 11, 9, 5, 3, 8},
{4, 1, 14, 8, 13, 6, 2, 11, 15, 12, 9, 7, 3, 10, 5, 0 },
{15, 12, 8, 2, 4, 9, 1, 7, 5, 11, 3, 14, 10, 0, 6, 13 } }, 
	{ {15, 1, 8, 14, 6, 11, 3, 4, 9, 7, 2, 13, 12, 0, 5, 10},
{3, 13, 4, 7, 15, 2, 8, 14, 12, 0, 1, 10, 6, 9, 11, 5 },
{0, 14, 7, 11, 10, 4, 13, 1, 5, 8, 12, 6, 9, 3, 2, 15 },
{13, 8, 10, 1, 3, 15, 4, 2, 11, 6, 7, 12, 0, 5, 14, 9 } }, 
	{ {10, 0, 9, 14, 6, 3, 15, 5, 1, 13, 12, 7, 11, 4, 2, 8},
{13, 7, 0, 9, 3, 4, 6, 10, 2, 8, 5, 14, 12, 11, 15, 1 },
{13, 6, 4, 9, 8, 15, 3, 0, 11, 1, 2, 12, 5, 10, 14, 7 },
{1, 10, 13, 0, 6, 9, 8, 7, 4, 15, 14, 3, 11, 5, 2, 12 } }, 
	{ {7, 13, 14, 3, 0, 6, 9, 10, 1, 2, 8, 5, 11, 12, 4, 15},
{13, 8, 11, 5, 6, 15, 0, 3, 4, 7, 2, 12, 1, 10, 14, 9},
{10, 6, 9, 0, 12, 11, 7, 13, 15, 1, 3, 14, 5, 2, 8, 4},
{3, 15, 0, 6, 10, 1, 13, 8, 9, 4, 5, 11, 12, 7, 2, 14} }, 
	{ {2, 12, 4, 1, 7, 10, 11, 6, 8, 5, 3, 15, 13, 0, 14, 9},
{14, 11, 2, 12, 4, 7, 13, 1, 5, 0, 15, 10, 3, 9, 8, 6},
{4, 2, 1, 11, 10, 13, 7, 8, 15, 9, 12, 5, 6, 3, 0, 14},
{11, 8, 12, 7, 1, 14, 2, 13, 6, 15, 0, 9, 10, 4, 5, 3} }, 
	{ {12, 1, 10, 15, 9, 2, 6, 8, 0, 13, 3, 4, 14, 7, 5, 11},
{10, 15, 4, 2, 7, 12, 9, 5, 6, 1, 13, 14, 0, 11, 3, 8 },
{9, 14, 15, 5, 2, 8, 12, 3, 7, 0, 4, 10, 1, 13, 11, 6 },
{4, 3, 2, 12, 9, 5, 15, 10, 11, 14, 1, 7, 6, 0, 8, 13 } },
	{ {4, 11, 2, 14, 15, 0, 8, 13, 3, 12, 9, 7, 5, 10, 6, 1},
{13, 0, 11, 7, 4, 9, 1, 10, 14, 3, 5, 12, 2, 15, 8, 6 },
{1, 4, 11, 13, 12, 3, 7, 14, 10, 15, 6, 8, 0, 5, 9, 2 },
{6, 11, 13, 8, 1, 4, 10, 7, 9, 5, 0, 15, 14, 2, 3, 12 } },
	{ {13, 2, 8, 4, 6, 15, 11, 1, 10, 9, 3, 14, 5, 0, 12, 7},
{1, 15, 13, 8, 10, 3, 7, 4, 12, 5, 6, 11, 0, 14, 9, 2 },
{7, 11, 4, 1, 9, 12, 14, 2, 0, 6, 10, 13, 15, 3, 5, 8 },
{2, 1, 14, 7, 4, 10, 8, 13, 15, 12, 9, 0, 3, 5, 6, 11 } }
};

int P[] ={15, 6, 19, 20,
	28, 11, 27, 16,
	0, 14, 22, 25,
	4, 17, 30, 9,
	1, 7, 23, 13,
	31, 26, 2, 8,
	18, 12, 29, 5,
	21, 10, 3, 24};

int IP_1[] = {
39, 7, 47, 15, 55, 23, 63, 31,
38, 6, 46, 14, 54, 22, 62, 30,
37, 5, 45, 13, 53, 21, 61, 29,
36, 4, 44, 12, 52, 20, 60, 28,
35, 3, 43, 11, 51, 19, 59, 27,
34, 2, 42, 10, 50, 18, 58, 26,
33, 1, 41, 9, 49, 17, 57, 25,
32, 0, 40, 8, 48, 16, 56, 24 };

void fun()
{
	for(int i = 0; i < 56; i++)
	{
		cout << PC_1[i] - 1 << ", ";

		if (!(i % 8))
			cout << "\n";
	}
}

template< typename T, size_t N, size_t M >
void printArray(T(&theArray)[N][M], int char_endl_nbr) {
	for (int x = 0; x < N; x++) 
	{
		for (int y = 0; y < M; y++)
		{
			cout << theArray[x][y];
			if (y == char_endl_nbr)
				cout << endl;
		}
		cout << endl;
	}
}


//template< typename T, size_t N, size_t M >
//void printArray2(T(&theArray)[N][M], int char_endl_nbr) {
//	for (int x = 0; x < N; x++)
//	{
//		for (int y = 0; y < M; y++)
//		{
//			cout << theArray[x][y];
//			if (y == char_endl_nbr)
//				cout << endl;
//			if (!(y % 6))
//				cout << " ";
//		}
//		cout << endl;
//	}
//}
template< typename T, size_t N, size_t M >
void printArray2(T(&theArray)[N][M], int char_endl_nbr) {
	for (int x = 0; x < N; x++)
	{
		for (int y = 0; y < M; y++)
		{
			cout << theArray[x][y];
			if (y == char_endl_nbr)
				cout << endl;
//			if (!(y % 6))
//				cout << " ";
		}
		cout << endl;
	}
}

//bitset bytesToBitset<int numBytes>(byte *bytes)
//{
//	std::bitset<numBytes * CHAR_BIT> b;
//
//	for (int i = 0; i < numBytes; ++i)
//	{
//		byte cur = bytes[i];
//		int offset = i * CHAR_BIT;
//
//		for (int bit = 0; bit < CHAR_BIT; ++bit)
//		{
//			b[offset] = cur & 1;
//			++offset;   // Move to next bit in b
//			cur >>= 1;  // Move to next bit in array
//		}
//	}
//
//	return b;
//}

//
//template<int numBytes>
//void bytesToBitset(string key_binary_ret)
//{
////	unsigned char c = 'a';
//
//	char const *c_key = key_binary_ret.c_str();
//
//	for (int i = 0; i < key_binary_ret.size(); i++)
//	{
//		for (int j = 0; j < 8; j++)
//		{
//
//			std::cout << ((c_key[i] >> j) & 1);
//		}
//		cout << " ";
//	}
//	
//}
//
//void bytesToBitset(string key_binary_ret)
//{
//	//	unsigned char c = 'a';
//
//	char const *c_key = key_binary_ret.c_str();
//
//	for (int i = 0; i < key_binary_ret.size(); i++)
//	{
//		for (int j = 0; j < 8; j++)
//		{
//
//			std::cout << ((c_key[i] >> j) & 1);
//		}
//		cout << " ";
//	}
//
//}
//
//int *get_bits(int n, int bitswanted) {
//	int *bits = (int *)malloc(sizeof(int) * bitswanted);
//
//	int k;
//	for (k = 0; k<bitswanted; k++) {
//		int mask = 1 << k;
//		int masked_n = n & mask;
//		int thebit = masked_n >> k;
//		bits[k] = thebit;
//	}
//
//	return bits;
//}
//

//template<int numBytes>
//bitset<numBytes * CHAR_BIT>bytesToBitset(char const *bytes)
//{
////	char const *bytes = key_binary_ret.c_str();
//	bitset<numBytes * CHAR_BIT> b = *bytes;
//
//	for (int i = 1; i < numBytes; ++i)
//	{
//		b <<= CHAR_BIT;  // Move to next bit in array
//		b |= bytes[i];    // Set the lowest CHAR_BIT bits
//	}
//
//	return b;
//}
//
//

void permutePC(int key_binary[], int key_binary_ret[], int key_binary_size, int PC[])
{
	for (int i = 0; i < key_binary_size; i++)
		key_binary_ret[i] = key_binary[PC[i]];

}

//C and D should have 28 array memebers
void createSubkeys(int key[], const int key_size, int C[], int D[], int CD_size, int run_number)
{
	const int size = key_size / 2;
	int tmp_C[28], tmp_D[28];
	for(int i = 0; i < key_size / 2; i++)
	{
		tmp_C[i] = key[i];
		tmp_D[i] = key[i + CD_size];
	}

	for(int i = 0; i < CD_size; i++)
	{
		C[i] = tmp_C[(i + shifts[run_number]) % CD_size];
		D[i] = tmp_D[(i + shifts[run_number]) % CD_size];
	}

}

void decimal2Binary(int decimal_int, int binary_int[], int run_number)
{
	int remainder;

	if (decimal_int <= 1) {
		binary_int[run_number] = decimal_int;
		return;
	}

	remainder = decimal_int % 2;
	decimal2Binary(decimal_int >> 1, binary_int, run_number + 1);
	binary_int[run_number] = remainder;
}

void reverseTab(int tab[], int tab_length)
{
	for (int i = 0; i < tab_length / 2; i++)
	{
		int tmp = tab[i];
		tab[i] = tab[tab_length - i - 1];
		tab[tab_length - i - 1] = tmp;

	}
}

void appendKeys(int leftKey[], int rightKey[], int key_size, int key_ret[])
{
	for(int i = 0; i < key_size; i++)
	{
		key_ret[i] = leftKey[i];
		key_ret[i + key_size] = rightKey[i];
	}
}


void expand(int R[], int tab_ret[], int E[], int E_size)
{

	for (int i = 0; i < E_size; i++)
		tab_ret[i] = R[E[i]];
}

void xor(int first_tab[], int second_tab[], int tab_size, int tab_ret[])
{
	for (int i = 0; i < tab_size; i++)
		tab_ret[i] = (int)(!first_tab[i] != !second_tab[i]);

}

//-->
long long binary2Decimal(int binary_int[], int tab_length)
{
	string int_string = "";

	for (int i = 0; i < tab_length; i++)
		int_string += to_string(binary_int[i]);
	stringstream ss;
	ss << int_string;
	string str = ss.str();
	unsigned long long value = std::stoull(str, 0, 2);
	//std::cout << value << std::endl;
	return value;
}

void f(int R[], int K[], int ret_tab[])
{
	int R_expanded[48];
	expand(R, R_expanded, E, 48);
	//DEBUG
//	for (int i = 0; i < 48; i++)
//	{
//		if (!(i % 6))
//			cout << " ";
//		cout << R_expanded[i];
//	}
//	cout << endl << endl << endl;
	
	int xored[48];
	xor (K, R_expanded, 48, xored);
	//DEBUG
//	for(int i = 0; i < 48; i++)
//	{
//		if (!(i % 6))
//			cout << " ";
//		cout << xored[i];
//	}
//	cout << endl << endl << endl;


	for(int i = 0; i < 8; i++)
	{
		int row[4] = {0, 0, 0, 0 }, column[4] = {0, 0, 0, 0};
		row[3] = xored[6 * i + 5];
		row[2] = xored[6 * i];
		column[0] = xored[6 * i + 1];
		column[1] = xored[6 * i + 2];
		column[2] = xored[6 * i + 3];
		column[3] = xored[6 * i + 4];

		int chunk_length = 4;
		int R_chunk[4] = {0, 0, 0, 0};
		decimal2Binary(S[i][binary2Decimal(row, 4)][binary2Decimal(column, 4)], R_chunk, 0);
		reverseTab(R_chunk, chunk_length);
		for(int j = 0; j < chunk_length; j++)
		{
			R[4 * i + j] = R_chunk[j];
		}

	}
	//DEBUG
//	for(int i = 0; i < 32; i++)
//	{
//		if (!(i % 4))
//			cout << " ";
//		cout << R[i];
//	}
//	cout << endl << endl << endl;
		
	permutePC(R, ret_tab, 32, P);
	//DEBUG
//	for (int i = 0; i < 32; i++)
//	{
//		if (!(i % 4))
//			cout << " ";
//		cout << ret_tab[i];
//	}
//	cout << endl << endl << endl;

}





void reverse(int L[], int R[], int tab_length, int ret_tab[])
{
	for (int i = 0; i < tab_length; i++)
	{
		ret_tab[i] = R[i];
		ret_tab[i + tab_length] = L[i];
	}
}

void messageEncode(int message_binary[], int message_size, int K[][48], int msg_ret[])
{
	int L[32], R[32];
	for(int i = 0; i < message_size / 2; i++)
	{
		L[i] = message_binary[i];
		R[i] = message_binary[i + message_size / 2];
	}

	int prev_L[32], prev_R[32];
	for (int i = 0; i < message_size / 2; i++)
	{
		prev_L[i] = L[i];
		prev_R[i] = R[i];
	}



	for(int i = 0; i < 16; i++)
	{	
		for (int j = 0; j < message_size / 2; j++)
			L[j] = prev_R[j];

		int tmp_f[32];
		f(prev_R, K[i], tmp_f);
		//DEBUG
//		for (int i = 0; i < 32; i++)
//		{
//			if (!(i % 4))
//				cout << " ";
//			cout << tmp_f[i];
//		}
//		cout << endl << endl << endl;

		xor(prev_L, tmp_f, 32, R);

		//DEBUG
//		for(int i = 0; i < 32; i++)
//		{
//			if (!(i % 4))
//				cout << " ";
//			cout << R[i];
//		}
//		cout << endl << endl << endl;

		//przepisanie R i L do prev_R i prev_L
		for(int i = 0; i < message_size / 2; i++)
		{
			prev_L[i] = L[i];
			prev_R[i] = R[i];
		}
	}



	int msg[64];
	reverse(L, R, 32, msg);
	//DEBUG
//	for (int i = 0; i < 64; i++)
//	{
//		if (!(i % 8))
//			cout << " ";
//		cout << msg[i];
//	}

	
	permutePC(msg, msg_ret, 64, IP_1);
	//DEBUG
//	for(int i = 0; i < 64; i++)
//	{
//		if (!(i % 8))
//			cout << " ";
//		cout << msg_ret[i];
//	}	


}	 





//key_binary_ret should be 64 bit long
void desEncyption(int message_binary[], int message_size,int key_binary[], int key_size, int msg_ret[])
{
	int des_block_size_bytes = 8;
	int des_block_size_bits = 64;

//	cout << "omg";
	//DEBUG
//		cout << message.size();
//		cout << "\n" << message << "\n";
//if (message.size() * CHAR_BIT != des_block_size_bits)
//		cout << message.size() * CHAR_BIT;
		

	if (message_size % des_block_size_bytes)
	{
		//int tmp_message_binary[message_size + des_block_size_bytes - (message_size % des_block_size_bytes)]
			cout << "KICIA";
	//	message_binary.append(des_block_size_bytes - (message.size() % des_block_size_bytes), '0');//mayby another char to append  
	}

	//OLD Verwsion with message as string 
//	if (message.size() % des_block_size_bytes)
//		message.append(des_block_size_bytes - (message.size() % des_block_size_bytes), '0');//mayby another char to append  

	//DEBUG
//		cout << "\n" << message << "\n";
//		cout << message.size();

	int key_binary_ret[56];
	permutePC(key_binary, key_binary_ret, sizeof(key_binary_ret) / sizeof(key_binary_ret[0]), PC_1);

	//DEBUG
//	for (int i = 0; i < 56; i++)
//	{
//		if (!(i % 7))
//			cout << "\n";
//		cout << key_binary_ret[i];
//	}

	int subkeys_number = 17;
	int subkey_size = 28;
	int subkeys[17][56];
	int C[28], D[28];

	for (int i = 0; i < 56; i++)
		subkeys[0][i] = key_binary_ret[i];

	for(int i = 0; i < subkeys_number - 1; i++)
	{
		createSubkeys(subkeys[i], sizeof(key_binary_ret) / sizeof(key_binary_ret[0]), C, D, sizeof(C) / sizeof(C[0]), i);
		appendKeys(C, D, subkey_size, subkeys[i + 1]);
		//DEBUG
//		for(int i = 0; i < subkeys_number; i++)
//			for(int j = 0; j < 56; j++)
//				cout << 
//		for (int i = 0; i < 28; i++)
//		{
//			cout << C[i];
//		}
//		cout << endl;
//		for (int i = 0; i < 28; i++)
//		{
//			cout << D[i];
//		}
//		cout << endl;

	}

	//DEBUG
//	printArray(subkeys, 1000);

	int K[16][48];
	for(int i = 0; i < 16; i++)
	{
		permutePC(subkeys[i + 1], K[i], sizeof(K[0]) / sizeof(K[0][0]), PC_2);
	}

	//DEBUG
//	printArray2(K, 10000);

	//WARNING!!! message size 
	int message_binary_ret[64];
	permutePC(message_binary, message_binary_ret, message_size, IP);

	//DEBUG
//	for(int i = 0; i < message_size; i++)
//		cout << message_binary_ret[i];
	messageEncode(message_binary_ret, message_size, K, msg_ret);

}


void bytes2Bits(vector<BYTE> bytes, int bits[])
{
//	while(bytes.begin())
//	{
//		
//	}
	for(int i = 0; i < bytes.size(); i++)
	{
		BYTE cur = bytes[i];
		int offset = i * CHAR_BIT;

		for (int bit = 0; bit < CHAR_BIT; bit++, offset++)
		{
			bits[offset] = cur & 1;
			cur >>= 1;  // Move to next bit in array
		}
	}

	/////
//	std::bitset<numBytes * CHAR_BIT> b;
//
//	for (int i = 0; i < numBytes; ++i)
//	{
//		BYTE cur = bytes[i];
//		int offset = i * CHAR_BIT;
//
//		for (int bit = 0; bit < CHAR_BIT; ++bit)
//		{
//			b[offset] = cur & 1;
//			++offset;   // Move to next bit in b
//			cur >>= 1;  // Move to next bit in array
//		}
//	}

}

vector<BYTE> hex2Byte(string string_hex)
{
	stringstream converter;
	istringstream istringstream_hex(string_hex);
	vector<BYTE> bytes;

	string word;
	while (istringstream_hex >> word)
	{
		BYTE temp;
		converter << std::hex << word;
		converter >> temp;
		bytes.push_back(temp);
	}

	return bytes;
}

enum DesStringBase
{
	Decimal, 
	Hex,
	Binary//not implemented 
};

const char* hex_char_to_bin(char c)
{
	// TODO handle default / error
	switch (toupper(c))
	{
	case '0': return "0000";
	case '1': return "0001";
	case '2': return "0010";
	case '3': return "0011";
	case '4': return "0100";
	case '5': return "0101";
	case '6': return "0110";
	case '7': return "0111";
	case '8': return "1000";
	case '9': return "1001";
	case 'A': return "1010";
	case 'B': return "1011";
	case 'C': return "1100";
	case 'D': return "1101";
	case 'E': return "1110";
	case 'F': return "1111";
	}
}

std::string hex2Bin(const std::string& hex)
{
	// TODO use a loop from <algorithm> or smth
	std::string bin;
	for (unsigned i = 0; i != hex.length(); ++i)
		bin += hex_char_to_bin(hex[i]);
	return bin;
}

vector<int> str2Int(string& str_int)
{
	vector<int> int_vector;
	for (int i = 0; i < str_int.size(); i++)
		int_vector.push_back(str_int.c_str()[i] - '0');

	return int_vector;
}

void bin2Hex(string binary)
{
	long int longint = 0;
	for (int i = 0; i < binary.size(); i++)
		longint += (binary[binary.size() - i - 1] - 48) * pow(2, i);
	cout << setbase(16);
	cout << longint;

}

string getHexStringFromBinaryString(string sHex)
{
	string sReturn = "";
	int bit_length = 4;
	const string const bins[] = { "0000", "0001", "0010", "0011", "0100", "0101", "0110", "0111",
		"1000", "1001", "1010", "1011", "1100", "1101", "1110", "1111" };
	for (int i = 0; i < sHex.length() / bit_length; ++i)
	{
		string s = sHex.substr(bit_length * i, bit_length);

		if(s == bins[0])
			 sReturn.append("0");
		if (s == bins[1])
			 sReturn.append("1");
		if (s == bins[2])
			sReturn.append("2");
		if (s == bins[3])
			sReturn.append("3");
		if (s == bins[4])
			sReturn.append("4");
		if (s == bins[5])
			 sReturn.append("5");
		if (s == bins[6])
			 sReturn.append("6");
		if (s == bins[7])
			sReturn.append("7");
		if (s == bins[8])
			sReturn.append("8");
		if (s == bins[9])
			sReturn.append("9");
		if (s == bins[10])
			sReturn.append("A");
		if (s == bins[11])
			sReturn.append("B");
		if (s == bins[12])
			 sReturn.append("C");
		if (s == bins[13])
			 sReturn.append("D");
		if (s == bins[14])
			sReturn.append("E");
		if (s == bins[15])
			 sReturn.append("F");
		}

	return sReturn;

}


string desEncyption(string message2Encrypt, string key, DesStringBase base)
{
	//TODO implement different bases
	string str_message = hex2Bin(message2Encrypt);
	vector<int> message_binary = str2Int(str_message);
	string str_key = hex2Bin(key);
	vector<int> key_binary = str2Int(str_key);

	if(base == Decimal)
	{
		//TODO implement decimal to hex
	}

//	int message_binary[] = { 0,0,0,0, 0,0,0,1, 0,0,1,0, 0,0,1,1, 0,1,0,0, 0,1,0,1, 0,1,1,0, 0,1,1,1, 1,0,0,0, 1,0,0,1, 1,0,1,0, 1,0,1,1, 1,1,0,0, 1,1,0,1, 1,1,1,0, 1,1,1,1 };
//	int key_binary[] = { 0,0,0,1,0,0,1,1, 0,0,1,1,0,1,0,0, 0,1,0,1,0,1,1,1, 0,1,1,1,1,0,0,1, 1,0,0,1,1,0,1,1, 1,0,1,1,1,1,0,0, 1,1,0,1,1,1,1,1, 1,1,1,1,0,0,0,1 };
	int msg_ret[64];
	desEncyption(&message_binary[0], message_binary.size(), &key_binary[0], key.size(), msg_ret);
	//DEBUG
	//	for (int i = 0; i < 64; i++)
	//	{
	//		if (!(i % 8))
	//			cout << " ";
	//		cout << msg_ret[i];
	//	}

	string binary;
	for (int i = 0; i < 64; i++)
		binary.push_back(std::to_string(msg_ret[i]).c_str()[0]);
	//DEBUG
	//cout << binary;

	cout << getHexStringFromBinaryString(binary);
	
	return "NOT IMPLEMENTED";
}


//Shifts
//11110000110011001010101011110101010101100110011110001111
//11100001100110010101010111111010101011001100111100011110
//11000011001100101010101111110101010110011001111000111101
//00001100110010101010111111110101011001100111100011110101
//00110011001010101011111111000101100110011110001111010101
//11001100101010101111111100000110011001111000111101010101
//00110010101010111111110000111001100111100011110101010101
//11001010101011111111000011000110011110001111010101010110
//00101010101111111100001100111001111000111101010101011001
//01010101011111111000011001100011110001111010101010110011
//01010101111111100001100110011111000111101010101011001100
//01010111111110000110011001011100011110101010101100110011
//01011111111000011001100101010001111010101010110011001111
//01111111100001100110010101010111101010101011001100111100
//11111110000110011001010101011110101010101100110011110001
//11111000011001100101010101111010101010110011001111000111
//11110000110011001010101011110101010101100110011110001111

//K
//000110110000001011101111111111000111000001110010
//011110011010111011011001110110111100100111100101
//010101011111110010001010010000101100111110011001
//011100101010110111010110110110110011010100011101
//011111001110110000000111111010110101001110101000
//011000111010010100111110010100000111101100101111
//111011001000010010110111111101100001100010111100
//111101111000101000111010110000010011101111111011
//111000001101101111101011111011011110011110000001
//101100011111001101000111101110100100011001001111
//001000010101111111010011110111101101001110000110
//011101010111000111110101100101000110011111101001
//100101111100010111010001111110101011101001000001
//010111110100001110110111111100101110011100111010
//101111111001000110001101001111010011111100001010
//110010110011110110001011000011100001011111110101


//MESSAGE AFTER IP
//1100110000000000110011001111111111110000101010101111000010101010






int main()
{
	//TESTING
//	char msg[] = { 'a' , '\0'};
//	cout << msg << endl;
//	cout << sizeof(msg) << endl;
//	cout << sizeof(msg) * CHAR_BIT << endl;
//	
//	string str = "a";
//	cout << endl << str << endl;
//	cout << str.size() << endl;
//	cout << str.size() * CHAR_BIT << endl;
//
//	
//	int a = 0x0123456789ABCDEF;
//	stringstream stream;
//	stream << hex << a;
//	string str2 = stream.str();
//	cout << endl << str2 << endl;
//	cout << hex << str2 << endl;
//	cout << str2.size() << endl;
//	cout << str2.size() * CHAR_BIT << endl;
	//TESTING
//	int binary_int[4];
//	decimal2Binary(13, binary_int, 0);
//	reverseTab(binary_int, 4);
//	cout << binary_int[0] << binary_int[1] << binary_int[2] << binary_int[3] << endl;



//	int decimal_int = 13;
//	stringstream string_hex;
//	string_hex << decimal_int;
//	string str = string_hex.str();
//	unsigned long long value = std::stoull(str, 0, 10);
//	cout << value << endl;
//	void decimal2Binary(int decimal_int) {
//		int remainder;
//
//		if (decimal_int <= 1) {
//			cout << decimal_int;
//			return;
//		}
//
//		remainder = decimal_int % 2;
//		decimal2Binary(decimal_int >> 1);
//		cout << remainder;
//	}

//	int row[] = { 0,0,0, 1 }, column[] = { 1, 1, 0, 1 };
//	int R[4];
//	int chunk_length = 4;
//	int R_chunk[4] = { 0, 0, 0, 0 };
//	decimal2Binary(S[0][binary2Decimal(row, 4)][binary2Decimal(column, 4)], R_chunk, 0);
//	reverseTab(R_chunk, chunk_length);
//	for (int j = 0; j < chunk_length; j++)
//	{
//		R[ j] = R_chunk[j];
//	}
//
//	cout << R[0] << R[1] << R[2] << R[3] << endl;
//
//	string message = "0123456789ABCDEF";
//	string string_hex("01 23 45 67 89 AB CD EF");
////	vector<BYTE> bytes = hex2Byte(string_hex);
////	
//////	for (int i = 0; i < bytes.size(); i++)
//////	{
//////		if (!(i % 4))
//////			cout << " ";
//////		cout << bytes[i];
//////	}
////
////
////	int bits[8 * CHAR_BIT];	
////	bytes2Bits(bytes, bits);
////
////	for(int i = 0; i < 8 * CHAR_BIT / 2; i++)
////	{
////		if (!(i % 4))
////			cout << " ";
////		cout << bits[i];
////	}



	string message = "0123456789ABCDEF", key = "133457799BBCDFF1";
	int message_binary[] = { 0,0,0,0, 0,0,0,1, 0,0,1,0, 0,0,1,1, 0,1,0,0, 0,1,0,1, 0,1,1,0, 0,1,1,1, 1,0,0,0, 1,0,0,1, 1,0,1,0, 1,0,1,1, 1,1,0,0, 1,1,0,1, 1,1,1,0, 1,1,1,1};
	int key_binary[] = { 0,0,0,1,0,0,1,1, 0,0,1,1,0,1,0,0, 0,1,0,1,0,1,1,1, 0,1,1,1,1,0,0,1, 1,0,0,1,1,0,1,1, 1,0,1,1,1,1,0,0, 1,1,0,1,1,1,1,1, 1,1,1,1,0,0,0,1 };
	string cypherText = desEncyption(message, key, DesStringBase::Hex);


	//OLD
//	int key = 0x133457799BBCDFF1;
///	int* bits = get_bits(key, sizeof(key) * CHAR_BIT);
///	
///	int cntr = 0;
///	while(bits[cntr])
///	{
///		cout << bits[cntr++];
///		if (!(cntr % 8))
///			cout << " ";
///	}
//	
//	//	bytesToBitset(key);
//
////	stringstream string_hex;
///	string_hex << key;
///	string test = "0";
///	bytesToBitset<16>(string_hex.str());
///	desEncyption(message, key_binary_ret);
//	
////	char const *c_key = key_binary_ret.c_str();
///	int c_key_size = 0;
///	while (c_key[c_key_size])
///	{
///		c_key_size++;
///	}
///
///	cout << c_key_size;
//
//
////	cout << CHAR_BIT;
//
//	cout << key_binary_ret.size();
//	bitset<17 * CHAR_BIT> bits = bytesToBitset<17>(key_binary_ret.c_str());
//
//	for (int i = 0; i < bits.count(); i++)
//	{
//		if (!(i % 8))
//			cout << " ";
//		cout << bits[i];
//	}
	//OLD




	return 0;

}



////////
////////
////////
//#include <string>
//#include <bitset>
//#include <type_traits>
//
//// SFINAE for safety. Sue me for putting it in a macro for brevity on the function
//#define IS_INTEGRAL(T) typename std::enable_if< std::is_integral<T>::value >::type* = 0
//
////template<class T>
////std::string integral_to_binary_string(T byte, IS_INTEGRAL(T))
////{
////	std::bitset<sizeof(T) * CHAR_BIT> bs(byte);
////	return bs.to_string();
////}
//
//template<class T>
//std::string integral_to_binary_string(T* byte, IS_INTEGRAL(T))
//{
//	std::bitset<sizeof(T) * CHAR_BIT> map[16];  // each bitset has all 64 bits set to 0
//
//	for (int i = 0; i < 8; i++)
//	{
//		std::bitset<sizeof(unsigned char) * CHAR_BIT> bs(byte[i]);
//		cout << bs.to_string();
//		if (!(i % 2))
//			cout << " ";
//	}
//	
//	return "";
//}
//
//int main() {
//	unsigned char byte = 0x133457799BBCDFF1; // 0000 0011
////	unsigned char byte_array[] = { 0x1, 0x3, 0x3, 0x4, 0x5, 0x7, 0x7, 0x9, 0x9, 0xB, 0xB, 0xC, 0xD, 0xF, 0xF, 0x1 };
//	unsigned char byte_array[] = { 0x13, 0x34, 0x57, 0x79, 0x9B, 0xBC, 0xDF, 0xF1};
//
//	std::cout << integral_to_binary_string(byte_array);
//	//std::cin.get();
//}




//int binary2Decimal(int binary_int[])
//{
//	if (binary_int[0] == 0 && binary_int[1] == 0 && binary_int[2] == 0 && binary_int[3] == 0)
//		return ;
//	else if (binary_int[0] == && binary_int[1] == && binary_int[2] == && binary_int[3] == )
//		return ;
//	else if (binary_int[0] == && binary_int[1] == && binary_int[2] == && binary_int[3] == )
//		return ;
//	else if (binary_int[0] == && binary_int[1] == && binary_int[2] == && binary_int[3] == )
//		return ;
//	else if (binary_int[0] == && binary_int[1] == && binary_int[2] == && binary_int[3] == )
//		return ;
//	else if (binary_int[0] == && binary_int[1] == && binary_int[2] == && binary_int[3] == )
//		return ;
//	else if (binary_int[0] == && binary_int[1] == && binary_int[2] == && binary_int[3] == )
//		return ;
//	else if (binary_int[0] == && binary_int[1] == && binary_int[2] == && binary_int[3] == )
//		return ;
//	else if (binary_int[0] == && binary_int[1] == && binary_int[2] == && binary_int[3] == )
//		return ;
//	else if (binary_int[0] == && binary_int[1] == && binary_int[2] == && binary_int[3] == )
//		return ;
//	else if (binary_int[0] == && binary_int[1] == && binary_int[2] == && binary_int[3] == )
//		return ;
//	else if (binary_int[0] == && binary_int[1] == && binary_int[2] == && binary_int[3] == )
//		return ;
//	else if (binary_int[0] == && binary_int[1] == && binary_int[2] == && binary_int[3] == )
//		return ;
//	else if (binary_int[0] == && binary_int[1] == && binary_int[2] == && binary_int[3] == )
//		return ;
//	else if (binary_int[0] == && binary_int[1] == && binary_int[2] == && binary_int[3] == )
//		return ;
//	else if (binary_int[0] == && binary_int[1] == && binary_int[2] == && binary_int[3] == )
//		return ;
//	else if (binary_int[0] == && binary_int[1] == && binary_int[2] == && binary_int[3] == )
//		return;
//
//		return -1000500;
//}
//














////////////////////////////////////
//
//7 13 14 3 0 6 9 10 1 2 8 5 11 12 4 15
//13 8 11 5 6 15 0 3 4 7 2 12 1 10 14 9
//10 6 9 0 12 11 7 13 15 1 3 14 5 2 8 4
//3 15 0 6 10 1 13 8 9 4 5 11 12 7 2 14
//2 12 4 1 7 10 11 6 8 5 3 15 13 0 14 9
//14 11 2 12 4 7 13 1 5 0 15 10 3 9 8 6
//4 2 1 11 10 13 7 8 15 9 12 5 6 3 0 14
//11 8 12 7 1 14 2 13 6 15 0 9 10 4 5 3
//12 1 10 15 9 2 6 8 0 13 3 4 14 7 5 11
//10 15 4 2 7 12 9 5 6 1 13 14 0 11 3 8
//9 14 15 5 2 8 12 3 7 0 4 10 1 13 11 6
//4 3 2 12 9 5 15 10 11 14 1 7 6 0 8 13
//4 11 2 14 15 0 8 13 3 12 9 7 5 10 6 1
//13 0 11 7 4 9 1 10 14 3 5 12 2 15 8 6
//1 4 11 13 12 3 7 14 10 15 6 8 0 5 9 2
//6 11 13 8 1 4 10 7 9 5 0 15 14 2 3 12
//13 2 8 4 6 15 11 1 10 9 3 14 5 0 12 7
//1 15 13 8 10 3 7 4 12 5 6 11 0 14 9 2
//7 11 4 1 9 12 14 2 0 6 10 13 15 3 5 8
//2 1 14 7 4 10 8 13 15 12 9 0 3 5 6 11




//
//7 13 14 3 0 6 9 10 1 2 8 5 11 12 4 15
//13 8 11 5 6 15 0 3 4 7 2 12 1 10 14 9
//10 6 9 0 12 11 7 13 15 1 3 14 5 2 8 4
//3 15 0 6 10 1 13 8 9 4 5 11 12 7 2 14
//2 12 4 1 7 10 11 6 8 5 3 15 13 0 14 9
//14 11 2 12 4 7 13 1 5 0 15 10 3 9 8 6
//4 2 1 11 10 13 7 8 15 9 12 5 6 3 0 14
//11 8 12 7 1 14 2 13 6 15 0 9 10 4 5 3
//12 1 10 15 9 2 6 8 0 13 3 4 14 7 5 11
//10 15 4 2 7 12 9 5 6 1 13 14 0 11 3 8
//9 14 15 5 2 8 12 3 7 0 4 10 1 13 11 6
//4 3 2 12 9 5 15 10 11 14 1 7 6 0 8 13
//4 11 2 14 15 0 8 13 3 12 9 7 5 10 6 1
//13 0 11 7 4 9 1 10 14 3 5 12 2 15 8 6
//1 4 11 13 12 3 7 14 10 15 6 8 0 5 9 2
//6 11 13 8 1 4 10 7 9 5 0 15 14 2 3 12
//13 2 8 4 6 15 11 1 10 9 3 14 5 0 12 7
//1 15 13 8 10 3 7 4 12 5 6 11 0 14 9 2
//7 11 4 1 9 12 14 2 0 6 10 13 15 3 5 8
//2 1 14 7 4 10 8 13 15 12 9 0 3 5 6 11