#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>
#include <stdlib.h>
#include <vector>
#include <iomanip>

#include "hip/hip_runtime.h"
#include ""

#include "kernel.cuh"
#include "Utils.h"
#include "BinaryUtils.cuh"
#include "Arrays.h"

typedef unsigned char BYTE;

using namespace std;

__device__ int key_cracked = 0;

__constant__ int d_PC_1[56], d_shifts[16], d_PC_2[48], d_IP[64], d_E[48], d_S[8][4][16], d_P[32], d_IP_1[64];

__device__ void permutePC(int key_binary[], int key_binary_ret[], int key_binary_size, const int PC[])
{
	for (int i = 0; i < key_binary_size; i++)
		key_binary_ret[i] = key_binary[PC[i]];

}

//C and D should have 28 array memebers PROBABLY!!!
__device__ void createSubkeys(int key[], const int key_size, int C[], int D[], int CD_size, int run_number)
{
	const int size = key_size / 2;
	int tmp_C[28], tmp_D[28];
	for (int i = 0; i < key_size / 2; i++)
	{
		tmp_C[i] = key[i];
		tmp_D[i] = key[i + CD_size];
	}

	for (int i = 0; i < CD_size; i++)
	{
		C[i] = tmp_C[(i + d_shifts[run_number]) % CD_size];
		D[i] = tmp_D[(i + d_shifts[run_number]) % CD_size];
	}

}


__host__ __device__ void decimal2Binary(int decimal_int, int binary_int[], int run_number)
{
	if (decimal_int <= 1) {
		binary_int[run_number] = decimal_int;
		return;
	}

	int remainder = decimal_int % 2;
	decimal2Binary(decimal_int >> 1, binary_int, run_number + 1);
	binary_int[run_number] = remainder;
}


__device__ void reverseTab(int tab[], int tab_length)
{
	for (int i = 0; i < tab_length / 2; i++)
	{
		int tmp = tab[i];
		tab[i] = tab[tab_length - i - 1];
		tab[tab_length - i - 1] = tmp;

	}
}


__device__ void appendKeys(int leftKey[], int rightKey[], int key_size, int key_ret[])
{
	for (int i = 0; i < key_size; i++)
	{
		key_ret[i] = leftKey[i];
		key_ret[i + key_size] = rightKey[i];
	}
}


__device__ void expand(int R[], int tab_ret[], const int E[], int E_size)
{

	for (int i = 0; i < E_size; i++)
		tab_ret[i] = R[E[i]];
}


__device__ void xorArray(int first_tab[], int second_tab[], int tab_size, int tab_ret[])
{
	for (int i = 0; i < tab_size; i++)
		tab_ret[i] = (int)(!first_tab[i] != !second_tab[i]);

}


__device__ long long binary2Decimal(int binary_int[], int tab_length)
{
	long long dec = 0;

	for (int i = 0; i < tab_length; ++i)
	{
		int bin = binary_int[i];
		if (bin) dec = dec * 2 + 1;
		else dec *= 2;

	}

	return dec;
}


__device__ void f(int R[], int K[], int ret_tab[])
{
	int R_expanded[48];
	expand(R, R_expanded, d_E, 48);
	//DEBUG
	//	for (int i = 0; i < 48; i++)
	//	{
	//		if (!(i % 6))
	//			cout << " ";
	//		cout << R_expanded[i];
	//	}
	//	cout << endl << endl << endl;

	int xored[48];
	xorArray(K, R_expanded, 48, xored);
	//DEBUG
	//	for(int i = 0; i < 48; i++)
	//	{
	//		if (!(i % 6))
	//			cout << " ";
	//		cout << xored[i];
	//	}
	//	cout << endl << endl << endl;


	for (int i = 0; i < 8; i++)
	{
		int row[4] = { 0, 0, 0, 0 }, column[4] = { 0, 0, 0, 0 };
		row[3] = xored[6 * i + 5];
		row[2] = xored[6 * i];
		column[0] = xored[6 * i + 1];
		column[1] = xored[6 * i + 2];
		column[2] = xored[6 * i + 3];
		column[3] = xored[6 * i + 4];

		int chunk_length = 4;
		int R_chunk[4] = { 0, 0, 0, 0 };
		decimal2Binary(d_S[i][binary2Decimal(row, 4)][binary2Decimal(column, 4)], R_chunk, 0);
		reverseTab(R_chunk, chunk_length);
		for (int j = 0; j < chunk_length; j++)
		{
			R[4 * i + j] = R_chunk[j];
		}

	}
	//DEBUG
	//	for(int i = 0; i < 32; i++)
	//	{
	//		if (!(i % 4))
	//			cout << " ";
	//		cout << R[i];
	//	}
	//	cout << endl << endl << endl;

	permutePC(R, ret_tab, 32, d_P);
	//DEBUG
	//	for (int i = 0; i < 32; i++)
	//	{
	//		if (!(i % 4))
	//			cout << " ";
	//		cout << ret_tab[i];
	//	}
	//	cout << endl << endl << endl;

}


__device__ void reverse(int L[], int R[], int tab_length, int ret_tab[])
{
	for (int i = 0; i < tab_length; i++)
	{
		ret_tab[i] = R[i];
		ret_tab[i + tab_length] = L[i];
	}
}


__device__ void messageEncode(int message_binary[], int message_size, int K[][48], int msg_ret[])
{
	int L[32], R[32];
	for (int i = 0; i < message_size / 2; i++)
	{
		L[i] = message_binary[i];
		R[i] = message_binary[i + message_size / 2];
	}

	int prev_L[32], prev_R[32];
	for (int i = 0; i < message_size / 2; i++)
	{
		prev_L[i] = L[i];
		prev_R[i] = R[i];
	}

	for (int i = 0; i < 16; i++)
	{
		for (int j = 0; j < message_size / 2; j++)
			L[j] = prev_R[j];

		int tmp_f[32];
		f(prev_R, K[i], tmp_f);
		//DEBUG
		//		for (int i = 0; i < 32; i++)
		//		{
		//			if (!(i % 4))
		//				cout << " ";
		//			cout << tmp_f[i];
		//		}
		//		cout << endl << endl << endl;

		xorArray(prev_L, tmp_f, 32, R);

		//DEBUG
		//		for(int i = 0; i < 32; i++)
		//		{
		//			if (!(i % 4))
		//				cout << " ";
		//			cout << R[i];
		//		}
		//		cout << endl << endl << endl;

		//przepisanie R i L do prev_R i prev_L
		for (int j = 0; j < message_size / 2; j++)
		{
			prev_L[j] = L[j];
			prev_R[j] = R[j];
		}
	}

	int msg[64];
	reverse(L, R, 32, msg);
	//DEBUG
	//	for (int i = 0; i < 64; i++)
	//	{
	//		if (!(i % 8))
	//			cout << " ";
	//		cout << msg[i];
	//	}

	permutePC(msg, msg_ret, 64, d_IP_1);
	//DEBUG
	//	for(int i = 0; i < 64; i++)
	//	{
	//		if (!(i % 8))
	//			cout << " ";
	//		cout << msg_ret[i];
	//	}	

}


//key_binary_ret should be 64 bit long
__device__ void desEncryption(int message_binary[], int message_size, int key_binary[], int key_size, int msg_ret[])
{
	int des_block_size_bytes = 8;
	int des_block_size_bits = 64;

	//DEBUG
	//	printf("\n%s\n", "__device__ desEncryptionForDataBlock ");
	//	printf("%s\n", "message_binary");
	//	for (int i = 0; i < message_size; ++i)
	//	{
	//		printf("%i", message_binary[i]);
	//	}
	//		cout << message.size();
	//		cout << "\n" << message << "\n";
	//if (message.size() * CHAR_BIT != des_block_size_bits)
	//		cout << message.size() * CHAR_BIT;


	if (message_size % des_block_size_bytes)
	{
		//int tmp_message_binary[message_size + des_block_size_bytes - (message_size % des_block_size_bytes)]
		printf("%s\n", "KICIA");
		//	message_binary.append(des_block_size_bytes - (message.size() % des_block_size_bytes), '0');//mayby another char to append  
	}

	//OLD Verwsion with message as string 
	//	if (message.size() % des_block_size_bytes)
	//		message.append(des_block_size_bytes - (message.size() % des_block_size_bytes), '0');//mayby another char to append  

	//DEBUG
	//		cout << "\n" << message << "\n";
	//		cout << message.size();

	int key_binary_ret[56];
	permutePC(key_binary, key_binary_ret, sizeof(key_binary_ret) / sizeof(key_binary_ret[0]), d_PC_1);

	//DEBUG
	//	for (int i = 0; i < 56; i++)
	//	{
	//		if (!(i % 7))
	//			cout << "\n";
	//		cout << key_binary_ret[i];
	//	}

	int subkeys_number = 17;
	int subkey_size = 28;
	int subkeys[17][56];
	int C[28], D[28];

	for (int i = 0; i < 56; i++)
		subkeys[0][i] = key_binary_ret[i];

	for (int i = 0; i < subkeys_number - 1; i++)
	{
		createSubkeys(subkeys[i], sizeof(key_binary_ret) / sizeof(key_binary_ret[0]), C, D, sizeof(C) / sizeof(C[0]), i);
		appendKeys(C, D, subkey_size, subkeys[i + 1]);
		//DEBUG
		//		for(int i = 0; i < subkeys_number; i++)
		//			for(int j = 0; j < 56; j++)
		//				cout << 
		//		for (int i = 0; i < 28; i++)
		//		{
		//			cout << C[i];
		//		}
		//		cout << endl;
		//		for (int i = 0; i < 28; i++)
		//		{
		//			cout << D[i];
		//		}
		//		cout << endl;

	}

	//DEBUG
	//	printArray(subkeys, 1000);

	int K[16][48];
	for (int i = 0; i < 16; i++)
	{
		permutePC(subkeys[i + 1], K[i], sizeof(K[0]) / sizeof(K[0][0]), d_PC_2);
	}

	//DEBUG
	//	printArray2(K, 10000);

	//WARNING!!! message size 
	int message_binary_ret[64];
	permutePC(message_binary, message_binary_ret, message_size, d_IP);

	//DEBUG
	//	for(int i = 0; i < message_size; i++)
	//		cout << message_binary_ret[i];
	messageEncode(message_binary_ret, message_size, K, msg_ret);

}


const char* hexChar2Bin(char c)
{
	// TODO handle default / error
	switch (toupper(c))
	{
		case '0': return "0000";
		case '1': return "0001";
		case '2': return "0010";
		case '3': return "0011";
		case '4': return "0100";
		case '5': return "0101";
		case '6': return "0110";
		case '7': return "0111";
		case '8': return "1000";
		case '9': return "1001";
		case 'A': return "1010";
		case 'B': return "1011";
		case 'C': return "1100";
		case 'D': return "1101";
		case 'E': return "1110";
		case 'F': return "1111";
		default:
			return "ERROR_hexChar2Bin";
	}
}


string hex2Bin(const string& hex)
{
	// TODO use a loop from <algorithm> or smth
	string bin;
	for (unsigned i = 0; i != hex.length(); ++i)
		bin += hexChar2Bin(hex[i]);
	return bin;
}


void str2Int(string& str_int, int ret_int[], int ret_int_size)
{
	for (int i = 0; i < ret_int_size; i++)
		ret_int[i] = (str_int.c_str()[i] - '0');

}


__host__ __device__ void consecutiveKeyGenerator(unsigned long long &present_key, int next_key_binary[], int next_key_binary_size)
{
	for (int i = 0; i < next_key_binary_size; i++)
		next_key_binary[i] = 0;
	decimal2Binary(present_key, next_key_binary, 0);
}


__host__ __device__ bool compareArrays(int message[], int cyphertext[])
{
	for (int i = 0; i < 64; i++)
	{
		if (message[i] != cyphertext[i])
			return false;
	}

	return true;
}


__global__
void crackDes(int message_binary[], int cyphertext_binary[], int message_binary_size, unsigned long long computation_size)
{
	//printf("%s\n", "__global__ crackDes");



	//DEBUG
	//	for (int i = 0; i < possible_key_binary_size; ++i)
	//	{
	//		printf("%i", possible_key_binary[i]);
	//	}

	int msg_ret[64];

	//	printf("%s\n", "BEFORE desEncryptionForDataBlock");

	int possible_key_binary_size = 56;
	int possible_key_binary[56];
	unsigned long long present_key = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned long long last_key = present_key + computation_size;

	//unsigned long long temp = present_key + 2147483648;
	for (unsigned long long i = present_key; i < last_key; i++)
	{
		if (key_cracked == 1)
		{
			//		printf("%i", key_cracked);
			return;
		}
		consecutiveKeyGenerator(i, possible_key_binary, possible_key_binary_size);
		//	printf("%s\n", "BEFORE desEncryptionForDataBlock");
		desEncryption(message_binary, message_binary_size, possible_key_binary, 16, msg_ret);
		//	printf("%s\n", "AFTER desEncryptionForDataBlock");

		if (compareArrays(msg_ret, cyphertext_binary))
		{
			key_cracked = 1;
			printf("%s", "USED KEY IS: ");
			for (int i = 0; i < possible_key_binary_size; i++)
				printf("%i", possible_key_binary[i]);
			printf("\n");
		}
	}
}


__host__
void crackDes(string message, string cyphertext)
{
	string str_message = hex2Bin(message);
	int h_message_binary_size = 64;
	int h_message_binary[64];
	str2Int(str_message, h_message_binary, h_message_binary_size);

	string str_cyphertext = hex2Bin(cyphertext);
	int h_cyphertext_binary_size = 64;
	int h_cyphertext_binary[64];
	str2Int(str_cyphertext, h_cyphertext_binary, h_cyphertext_binary_size);

	int* d_message_binary = 0;
	hipMalloc((void**)&d_message_binary, h_message_binary_size * sizeof(int));
	hipMemcpy(d_message_binary, h_message_binary, h_message_binary_size * sizeof(int), hipMemcpyHostToDevice);

	int* d_cyphertext_binary = 0;
	hipMalloc((void**)&d_cyphertext_binary, h_cyphertext_binary_size * sizeof(int));
	hipMemcpy(d_cyphertext_binary, h_cyphertext_binary, h_cyphertext_binary_size * sizeof(int), hipMemcpyHostToDevice);

	const int threads_per_block = 512;//FERMI //1024; //2^10
	const int nbr_of_block_in_one_dim = 8192; //2 ^ 13;
	const int test_nbr_of_block = 32768; //2 ^ 15
	unsigned long long computation_size = pow(2, 47) / (nbr_of_block_in_one_dim);
//	printf("%s\n", "__host__ crackDes BEFORE __device__ crackDes");
	crackDes<<<nbr_of_block_in_one_dim, threads_per_block>>>(d_message_binary, d_cyphertext_binary, h_message_binary_size, computation_size);
//	printf("%s\n", "__host__ crackDes AFTER __device__ crackDes");
	//DEBUG
	//	for (int i = 0; i < 64; i++)
	//	{
	//		if (!(i % 8))
	//			cout << " ";
	//		cout << msg_ret[i];
	//	}

	//	string binary;
	//	for (int i = 0; i < 64; i++)
	//		binary.push_back(std::to_string(msg_ret[i]).c_str()[0]);
	//DEBUG
	//cout << binary;

}


__global__
void desEncryption(int message_binary[], int key_binary[], int message_binary_size, int msg_ret[])
{
	//DEBUG
	//	printf("%s\n", "before DEBUG __global__ desEncryptionForDataBlock MESSAGE_BINARY");
	//	for (int i = 0; i < message_binary_size; ++i)
	//	{
	//		printf("%i", message_binary[i]);
	//	}
	//	printf("%s\n", "after DEBUG __global__ desEncryptionForDataBlock MESSAGE_BINARY");

	//int msg_ret[64];
	//	printf("%s\n", "BEFORE desEncryptionForDataBlock");							14 should be here
	desEncryption(message_binary, message_binary_size, key_binary, 16, msg_ret);
	//	printf("%s\n", "before DEBUG __global__ desEncryptionForDataBlock MSG_RET");
	//	for (int i = 0; i < 64; ++i)
	//	{
	//		printf("%i", 123123123);
	//	}
	//	printf("%s\n", "after DEBUG __global__ desEncryptionForDataBlock MSG_RET");

}


__host__
string desEncryptionForDataBlock(string message, string key)
{
	string str_message = hex2Bin(message);
	int h_message_binary_size = 64;
	int h_message_binary[64];
	str2Int(str_message, h_message_binary, h_message_binary_size);

	string str_key = hex2Bin(key);
	int h_key_binary_size = 56;
	int h_key_binary[56];
	str2Int(str_key, h_key_binary, h_key_binary_size);

	int* d_message_binary = 0;
	hipMalloc((void**)&d_message_binary, h_message_binary_size * sizeof(int));
	hipMemcpy(d_message_binary, h_message_binary, h_message_binary_size * sizeof(int), hipMemcpyHostToDevice);

	int* d_key_binary = 0;
	hipMalloc((void**)&d_key_binary, h_key_binary_size * sizeof(int));
	hipMemcpy(d_key_binary, h_key_binary, h_key_binary_size * sizeof(int), hipMemcpyHostToDevice);

	int* d_msg_ret;
	hipMalloc((void**)&d_msg_ret, 64 * sizeof(int));

	//DEBUG
	//	printf("%s\n", "before DEBUG __host__ desEncryptionForDataBlock");
	//	for (int i = 0; i < 64; ++i)
	//	{
	//		printf("%i", h_message_binary[i]);
	//	}
	//	printf("%s\n", "after DEBUG __host__ desEncryptionForDataBlock");

	desEncryption<<<1, 1 >>>(d_message_binary, d_key_binary, 64, d_msg_ret);

	hipDeviceSynchronize();

	int* h_msg_ret = (int*)malloc(64 * sizeof(int));
	hipMemcpy(h_msg_ret, d_msg_ret, 64 * sizeof(int), hipMemcpyDeviceToHost);
	//DEBUG
	//	printf("\n%s\n", "before DEBUG __host__ desEncryptionForDataBlock H_MSG_RET");
	//	for (int i = 0; i < 64; ++i)
	//	{
	//		printf("%i", h_msg_ret[i]);
	//	}
	//	printf("%s\n", "after DEBUG __host__ desEncryptionForDataBlock H_MSG_RET");


	string binary;
	for (int i = 0; i < 64; i++)
		binary.push_back(std::to_string(h_msg_ret[i]).c_str()[0]);
	//DEBUG
	//cout << binary;

	return getHexStringFromBinaryString(binary);

}


__host__
string desEncryption(string message, string key)
{
	int block_size = 16;
	string encryptedMessage = "";
	for (int i = 0; i < message.size() / block_size; ++i)
		encryptedMessage += desEncryptionForDataBlock(message.substr(i * block_size, block_size), key);

	return encryptedMessage;
}


void initArrays()
{
	hipMemcpyToSymbol(HIP_SYMBOL(d_PC_1), PC_1, PC_1_size * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_shifts), shifts, shifts_size * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_PC_2), PC_2, PC_2_size * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_IP), IP, IP_size * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_E), E, E_size * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_P), P, P_size * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_S), S, S_size_1 * S_size_2 * S_size_3 * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_IP_1), IP_1, IP_1_size * sizeof(int));

}


void resizeGPUHeap()
{
	size_t size_heap, size_stack;
	hipDeviceSetLimit(hipLimitMallocHeapSize, 20000000 * sizeof(double));
	hipDeviceSetLimit(hipLimitStackSize, 12928);
	hipDeviceGetLimit(&size_heap, hipLimitMallocHeapSize);
	hipDeviceGetLimit(&size_stack, hipLimitStackSize);
	printf("Heap size found to be %d; Stack size found to be %d\n", (int)size_heap, (int)size_stack);

}
