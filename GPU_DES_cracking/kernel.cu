
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <string>
#include <iostream>
#include <bitset>
#include <sstream>
#include <stdlib.h>

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}
//
//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    cudaError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // cudaDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = cudaDeviceReset();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//cudaError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    cudaError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = cudaSetDevice(0);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = cudaMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = cudaMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = cudaMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = cudaMemcpy(dev_a, a, size * sizeof(int), cudaMemcpyHostToDevice);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = cudaMemcpy(dev_b, b, size * sizeof(int), cudaMemcpyHostToDevice);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = cudaGetLastError();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // cudaDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = cudaDeviceSynchronize();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = cudaMemcpy(c, dev_c, size * sizeof(int), cudaMemcpyDeviceToHost);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    cudaFree(dev_c);
//    cudaFree(dev_a);
//    cudaFree(dev_b);
//    
//    return cudaStatus;
//}

////////////////////////////////////////////////////


int PC_1[56] = { 56, 48, 40, 32, 24, 16, 8, 0,
57, 49, 41, 33, 25, 17, 9, 1,
58, 50, 42, 34, 26, 18, 10, 2,
59, 51, 43, 35, 62, 54, 46, 38,
30, 22, 14, 6, 61, 53, 45, 37,
29, 21, 13, 5, 60, 52, 44, 36,
28, 20, 12, 4, 27, 19, 11, 3};


int shifts[] = { 1, 1, 2, 2, 2, 2, 2, 2, 1, 2, 2, 2, 2, 2, 2, 1 };


void fun()
{
	for(int i = 0; i < 56; i++)
	{
		cout << PC_1[i] - 1 << ", ";

		if (!(i % 8))
			cout << "\n";
	}
}

template< typename T, size_t N, size_t M >
void printArray(T(&theArray)[N][M]) {
	for (int x = 0; x < N; x++) {
		for (int y = 0; y < M; y++) {
			cout << theArray[x][y];
			if (y == 27)
				cout << endl;
		}
		cout << endl;
	}
}


//bitset bytesToBitset<int numBytes>(byte *data)
//{
//	std::bitset<numBytes * CHAR_BIT> b;
//
//	for (int i = 0; i < numBytes; ++i)
//	{
//		byte cur = data[i];
//		int offset = i * CHAR_BIT;
//
//		for (int bit = 0; bit < CHAR_BIT; ++bit)
//		{
//			b[offset] = cur & 1;
//			++offset;   // Move to next bit in b
//			cur >>= 1;  // Move to next bit in array
//		}
//	}
//
//	return b;
//}

//
//template<int numBytes>
//void bytesToBitset(string key_binary_ret)
//{
////	unsigned char c = 'a';
//
//	char const *c_key = key_binary_ret.c_str();
//
//	for (int i = 0; i < key_binary_ret.size(); i++)
//	{
//		for (int j = 0; j < 8; j++)
//		{
//
//			std::cout << ((c_key[i] >> j) & 1);
//		}
//		cout << " ";
//	}
//	
//}
//
//void bytesToBitset(string key_binary_ret)
//{
//	//	unsigned char c = 'a';
//
//	char const *c_key = key_binary_ret.c_str();
//
//	for (int i = 0; i < key_binary_ret.size(); i++)
//	{
//		for (int j = 0; j < 8; j++)
//		{
//
//			std::cout << ((c_key[i] >> j) & 1);
//		}
//		cout << " ";
//	}
//
//}
//
//int *get_bits(int n, int bitswanted) {
//	int *bits = (int *)malloc(sizeof(int) * bitswanted);
//
//	int k;
//	for (k = 0; k<bitswanted; k++) {
//		int mask = 1 << k;
//		int masked_n = n & mask;
//		int thebit = masked_n >> k;
//		bits[k] = thebit;
//	}
//
//	return bits;
//}
//

//template<int numBytes>
//bitset<numBytes * CHAR_BIT>bytesToBitset(char const *data)
//{
////	char const *data = key_binary_ret.c_str();
//	bitset<numBytes * CHAR_BIT> b = *data;
//
//	for (int i = 1; i < numBytes; ++i)
//	{
//		b <<= CHAR_BIT;  // Move to next bit in array
//		b |= data[i];    // Set the lowest CHAR_BIT bits
//	}
//
//	return b;
//}
//
//

//key_binary_ret should be 64 bit long
void permutePC_1(int key_binary[], int key_binary_ret[], int key_binary_size)
{
	for (int i = 0; i < key_binary_size; i++)
		key_binary_ret[i] = key_binary[PC_1[i]];

}

//C and D should have 28 array memebers
void createSubkeys(int key[], const int key_size, int C[], int D[], int CD_size, int run_number)
{
	const int size = key_size / 2;
	int tmp_C[28], tmp_D[28];
	for(int i = 0; i < key_size / 2; i++)
	{
		tmp_C[i] = key[i];
		tmp_D[i] = key[i + CD_size];
	}

	for(int i = 0; i < CD_size; i++)
	{
		C[i] = tmp_C[(i + shifts[run_number]) % CD_size];
		D[i] = tmp_D[(i + shifts[run_number]) % CD_size];
	}

}

void appendKeys(int leftKey[], int rightKey[], int key_size, int key_ret[])
{
	for(int i = 0; i < key_size; i++)
	{
		key_ret[i] = leftKey[i];
		key_ret[i + key_size] = rightKey[i];
	}
}

//key_binary_ret should be 64 bit long
string desEncyption(string message, int key_binary[], int key_size)
{
	int des_block_size_bytes = 8;
	int des_block_size_bits = 64;

//	cout << "omg";
	//DEBUG
//		cout << message.size();
//		cout << "\n" << message << "\n";

	if (message.size() % des_block_size_bytes)
		message.append(des_block_size_bytes - (message.size() % des_block_size_bytes), '0');//mayby another char to append  

	//DEBUG
//		cout << "\n" << message << "\n";
//		cout << message.size();

	int key_binary_ret[56];
	permutePC_1(key_binary, key_binary_ret, sizeof(key_binary_ret) / sizeof(key_binary_ret[0]));

	//DEBUG
//	for (int i = 0; i < 56; i++)
//	{
//		if (!(i % 8))
//			cout << "\n";
//		cout << key_binary_ret[i];
//	}

	int subkeys_number = 17;
	int subkey_size = 28;
	int subkeys[17][56];
	int C[28], D[28];

	for (int i = 0; i < 56; i++)
		subkeys[0][i] = key_binary_ret[i];

	for(int i = 0; i < subkeys_number; i++)
	{
		createSubkeys(subkeys[i], sizeof(key_binary_ret) / sizeof(key_binary_ret[0]), C, D, sizeof(C) / sizeof(C[0]), i);
		appendKeys(C, D, subkey_size, subkeys[i + 1]);
		//DEBUG

//		for(int i = 0; i < subkeys_number; i++)
//			for(int j = 0; j < 56; j++)
//				cout << 
//		for (int i = 0; i < 28; i++)
//		{
//			cout << C[i];
//		}
//		cout << endl;
//		for (int i = 0; i < 28; i++)
//		{
//			cout << D[i];
//		}
//		cout << endl;

	}

	//DEBUG
	printArray(subkeys);


	return "NOT IMPLEMENTED";
}



//1111000011001100101010101111
//0101010101100110011110001111
//1110000110011001010101011111
//1010101011001100111100011110
//1100001100110010101010111111
//0101010110011001111000111101
//0000110011001010101011111111
//0101011001100111100011110101
//0011001100101010101111111100
//0101100110011110001111010101
//1100110010101010111111110000
//0110011001111000111101010101
//0011001010101011111111000011
//1001100111100011110101010101
//1100101010101111111100001100
//0110011110001111010101010110
//0010101010111111110000110011
//1001111000111101010101011001
//0101010101111111100001100110
//0011110001111010101010110011
//0101010111111110000110011001
//1111000111101010101011001100
//0101011111111000011001100101
//1100011110101010101100110011
//0101111111100001100110010101
//0001111010101010110011001111
//0111111110000110011001010101
//0111101010101011001100111100
//1111111000011001100101010101
//1110101010101100110011110001
//1111100001100110010101010111
//1010101010110011001111000111
//1111000011001100101010101111
//0101010101100110011110001111

int main()
{
	string message = "0123456789ABCDEF", key = "133457799BBCDFF1";
	int key_binary[] = { 0,0,0,1,0,0,1,1, 0,0,1,1,0,1,0,0, 0,1,0,1,0,1,1,1, 0,1,1,1,1,0,0,1, 1,0,0,1,1,0,1,1, 1,0,1,1,1,1,0,0, 1,1,0,1,1,1,1,1, 1,1,1,1,0,0,0,1 };
	string cypherText = desEncyption(message, key_binary, sizeof(key_binary) / sizeof(key_binary[0]));


	//OLD
//	int key_hex = 0x133457799BBCDFF1;
///	int* bits = get_bits(key_hex, sizeof(key_hex) * CHAR_BIT);
///	
///	int cntr = 0;
///	while(bits[cntr])
///	{
///		cout << bits[cntr++];
///		if (!(cntr % 8))
///			cout << " ";
///	}
//	
//	//	bytesToBitset(key_hex);
//
////	stringstream ss;
///	ss << key_hex;
///	string test = "0";
///	bytesToBitset<16>(ss.str());
///	desEncyption(message, key_binary_ret);
//	
////	char const *c_key = key_binary_ret.c_str();
///	int c_key_size = 0;
///	while (c_key[c_key_size])
///	{
///		c_key_size++;
///	}
///
///	cout << c_key_size;
//
//
////	cout << CHAR_BIT;
//
//	cout << key_binary_ret.size();
//	bitset<17 * CHAR_BIT> bits = bytesToBitset<17>(key_binary_ret.c_str());
//
//	for (int i = 0; i < bits.count(); i++)
//	{
//		if (!(i % 8))
//			cout << " ";
//		cout << bits[i];
//	}
	//OLD




	return 0;

}



////////
////////
////////
//#include <string>
//#include <bitset>
//#include <type_traits>
//
//// SFINAE for safety. Sue me for putting it in a macro for brevity on the function
//#define IS_INTEGRAL(T) typename std::enable_if< std::is_integral<T>::value >::type* = 0
//
////template<class T>
////std::string integral_to_binary_string(T byte, IS_INTEGRAL(T))
////{
////	std::bitset<sizeof(T) * CHAR_BIT> bs(byte);
////	return bs.to_string();
////}
//
//template<class T>
//std::string integral_to_binary_string(T* byte, IS_INTEGRAL(T))
//{
//	std::bitset<sizeof(T) * CHAR_BIT> map[16];  // each bitset has all 64 bits set to 0
//
//	for (int i = 0; i < 8; i++)
//	{
//		std::bitset<sizeof(unsigned char) * CHAR_BIT> bs(byte[i]);
//		cout << bs.to_string();
//		if (!(i % 2))
//			cout << " ";
//	}
//	
//	return "";
//}
//
//int main() {
//	unsigned char byte = 0x133457799BBCDFF1; // 0000 0011
////	unsigned char byte_array[] = { 0x1, 0x3, 0x3, 0x4, 0x5, 0x7, 0x7, 0x9, 0x9, 0xB, 0xB, 0xC, 0xD, 0xF, 0xF, 0x1 };
//	unsigned char byte_array[] = { 0x13, 0x34, 0x57, 0x79, 0x9B, 0xBC, 0xDF, 0xF1};
//
//	std::cout << integral_to_binary_string(byte_array);
//	//std::cin.get();
//}