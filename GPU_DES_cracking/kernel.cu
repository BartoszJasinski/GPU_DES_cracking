#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string>
#include <iostream>
#include <bitset>
#include <sstream>
#include <stdlib.h>
#include <vector>
#include <iomanip>

typedef unsigned char BYTE;

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


////////////////////////////////////////////////////
//int S1[4][16] = { { 14, 4, 13, 1, 2, 15, 11, 8, 3, 10, 6, 12, 5, 9, 0, 7 },
//{ 0, 15, 7, 4, 14, 2, 13, 1, 10, 6, 12, 11, 9, 5, 3, 8 },
//{ 4, 1, 14, 8, 13, 6, 2, 11, 15, 12, 9, 7, 3, 10, 5, 0 },
//{ 15, 12, 8, 2, 4, 9, 1, 7, 5, 11, 3, 14, 10, 0, 6, 13 } };
//
//int S2[4][16] = { { 15, 1, 8, 14, 6, 11, 3, 4, 9, 7, 2, 13, 12, 0, 5, 10 },
//{ 3, 13, 4, 7, 15, 2, 8, 14, 12, 0, 1, 10, 6, 9, 11, 5 },
//{ 0, 14, 7, 11, 10, 4, 13, 1, 5, 8, 12, 6, 9, 3, 2, 15, },
//{ 13, 8, 10, 1, 3, 15, 4, 2, 11, 6, 7, 12, 0, 5, 14, 9 } };
//
//int S3[4][16] = { { 10, 0, 9, 14, 6, 3, 15, 5, 1, 13, 12, 7, 11, 4, 2, 8 },
//{ 13, 7, 0, 9, 3, 4, 6, 10, 2, 8, 5, 14, 12, 11, 15, 1 },
//{ 13, 6, 4, 9, 8, 15, 3, 0, 11, 1, 2, 12, 5, 10, 14, 7 },
//{ 1, 10, 13, 0, 6, 9, 8, 7, 4, 15, 14, 3, 11, 5, 2, 12 } };
//
//int S4[4][16] = { { 7, 13, 14, 3, 0, 6, 9, 10, 1, 2, 8, 5, 11, 12, 4, 15 },
//{ 13, 8, 11, 5, 6, 15, 0, 3, 4, 7, 2, 12, 1, 10, 14, 9 },
//{ 10, 6, 9, 0, 12, 11, 7, 13, 15, 1, 3, 14, 5, 2, 8, 4 },
//{ 3, 15, 0, 6, 10, 1, 13, 8, 9, 4, 5, 11, 12, 7, 2, 14 } };
//
//int S5[4][16] = { { 2, 12, 4, 1, 7, 10, 11, 6, 8, 5, 3, 15, 13, 0, 14, 9 },
//{ 14, 11, 2, 12, 4, 7, 13, 1, 5, 0, 15, 10, 3, 9, 8, 6 },
//{ 4, 2, 1, 11, 10, 13, 7, 8, 15, 9, 12, 5, 6, 3, 0, 14 },
//{ 11, 8, 12, 7, 1, 14, 2, 13, 6, 15, 0, 9, 10, 4, 5, 3 } };
//
//int S6[4][16] = { { 12, 1, 10, 15, 9, 2, 6, 8, 0, 13, 3, 4, 14, 7, 5, 11 },
//{ 10, 15, 4, 2, 7, 12, 9, 5, 6, 1, 13, 14, 0, 11, 3, 8 },
//{ 9, 14, 15, 5, 2, 8, 12, 3, 7, 0, 4, 10, 1, 13, 11, 6 },
//{ 4, 3, 2, 12, 9, 5, 15, 10, 11, 14, 1, 7, 6, 0, 8, 13 } };
//
//int S7[4][16] = { { 4, 11, 2, 14, 15, 0, 8, 13, 3, 12, 9, 7, 5, 10, 6, 1 },
//{ 13, 0, 11, 7, 4, 9, 1, 10, 14, 3, 5, 12, 2, 15, 8, 6 },
//{ 1, 4, 11, 13, 12, 3, 7, 14, 10, 15, 6, 8, 0, 5, 9, 2 },
//{ 6, 11, 13, 8, 1, 4, 10, 7, 9, 5, 0, 15, 14, 2, 3, 12 } };
//
//int S8[4][16] = { { 13, 2, 8, 4, 6, 15, 11, 1, 10, 9, 3, 14, 5, 0, 12, 7 },
//{ 1, 15, 13, 8, 10, 3, 7, 4, 12, 5, 6, 11, 0, 14, 9, 2 },
//{ 7, 11, 4, 1, 9, 12, 14, 2, 0, 6, 10, 13, 15, 3, 5, 8 },
//{ 2, 1, 14, 7, 4, 10, 8, 13, 15, 12, 9, 0, 3, 5, 6, 11 } };


int PC_1_size = 56, shifts_size = 16, PC_2_size = 48, IP_size = 64, E_size = 48, S_size_1 = 8, S_size_2 = 4, S_size_3 = 16, P_size = 32, IP_1_size = 64;

//
//const int PC_1[56] = { 56, 48, 40, 32, 24, 16, 8, 0,
//57, 49, 41, 33, 25, 17, 9, 1,
//58, 50, 42, 34, 26, 18, 10, 2,
//59, 51, 43, 35, 62, 54, 46, 38,
//30, 22, 14, 6, 61, 53, 45, 37,
//29, 21, 13, 5, 60, 52, 44, 36,
//28, 20, 12, 4, 27, 19, 11, 3 };


const int PC_1[56] = { 49, 42, 35, 28, 21, 14, 7, 0,
50, 43, 36, 29, 22, 15, 8, 1,
51, 44, 37, 30, 23, 16, 9, 2,
52, 45, 38, 31, 55, 48, 41, 34,
27, 20, 13, 6, 54, 47, 40, 33,
26, 19, 12, 5, 53, 46, 39, 32,
25, 18, 11, 4, 24, 17, 10, 3};


const int shifts[16] = { 1, 1, 2, 2, 2, 2, 2, 2, 1, 2, 2, 2, 2, 2, 2, 1 };

const int PC_2[48] = { 13, 16, 10, 23, 0, 4,
2, 27, 14, 5, 20, 9,
22, 18, 11, 3, 25, 7,
15, 6, 26, 19, 12, 1,
40, 51, 30, 36, 46, 54,
29, 39, 50, 44, 32, 47,
43, 48, 38, 55, 33, 52,
45, 41, 49, 35, 28, 31 };


const int IP[64] = { 57, 49, 41, 33, 25, 17, 9, 1,
59, 51, 43, 35, 27, 19, 11, 3,
61, 53, 45, 37, 29, 21, 13, 5,
63, 55, 47, 39, 31, 23, 15, 7,
56, 48, 40, 32, 24, 16,  8, 0,
58, 50, 42, 34, 26, 18, 10, 2,
60, 52, 44, 36, 28, 20, 12, 4,
62, 54, 46, 38, 30, 22, 14, 6 };

const int E[48] = { 31, 0, 1, 2, 3, 4,
3, 4, 5, 6, 7, 8,
7, 8, 9, 10, 11, 12,
11, 12, 13, 14, 15, 16,
15, 16, 17, 18, 19, 20,
19, 20, 21, 22, 23, 24,
23, 24, 25, 26, 27, 28,
27, 28, 29, 30, 31, 0 };

const int S[8][4][16] = { { {14, 4, 13, 1, 2, 15, 11, 8, 3, 10, 6, 12, 5, 9, 0, 7},
{0, 15, 7, 4, 14, 2, 13, 1, 10, 6, 12, 11, 9, 5, 3, 8},
{4, 1, 14, 8, 13, 6, 2, 11, 15, 12, 9, 7, 3, 10, 5, 0 },
{15, 12, 8, 2, 4, 9, 1, 7, 5, 11, 3, 14, 10, 0, 6, 13 } }, 
	{ {15, 1, 8, 14, 6, 11, 3, 4, 9, 7, 2, 13, 12, 0, 5, 10},
{3, 13, 4, 7, 15, 2, 8, 14, 12, 0, 1, 10, 6, 9, 11, 5 },
{0, 14, 7, 11, 10, 4, 13, 1, 5, 8, 12, 6, 9, 3, 2, 15 },
{13, 8, 10, 1, 3, 15, 4, 2, 11, 6, 7, 12, 0, 5, 14, 9 } }, 
	{ {10, 0, 9, 14, 6, 3, 15, 5, 1, 13, 12, 7, 11, 4, 2, 8},
{13, 7, 0, 9, 3, 4, 6, 10, 2, 8, 5, 14, 12, 11, 15, 1 },
{13, 6, 4, 9, 8, 15, 3, 0, 11, 1, 2, 12, 5, 10, 14, 7 },
{1, 10, 13, 0, 6, 9, 8, 7, 4, 15, 14, 3, 11, 5, 2, 12 } }, 
	{ {7, 13, 14, 3, 0, 6, 9, 10, 1, 2, 8, 5, 11, 12, 4, 15},
{13, 8, 11, 5, 6, 15, 0, 3, 4, 7, 2, 12, 1, 10, 14, 9},
{10, 6, 9, 0, 12, 11, 7, 13, 15, 1, 3, 14, 5, 2, 8, 4},
{3, 15, 0, 6, 10, 1, 13, 8, 9, 4, 5, 11, 12, 7, 2, 14} }, 
	{ {2, 12, 4, 1, 7, 10, 11, 6, 8, 5, 3, 15, 13, 0, 14, 9},
{14, 11, 2, 12, 4, 7, 13, 1, 5, 0, 15, 10, 3, 9, 8, 6},
{4, 2, 1, 11, 10, 13, 7, 8, 15, 9, 12, 5, 6, 3, 0, 14},
{11, 8, 12, 7, 1, 14, 2, 13, 6, 15, 0, 9, 10, 4, 5, 3} }, 
	{ {12, 1, 10, 15, 9, 2, 6, 8, 0, 13, 3, 4, 14, 7, 5, 11},
{10, 15, 4, 2, 7, 12, 9, 5, 6, 1, 13, 14, 0, 11, 3, 8 },
{9, 14, 15, 5, 2, 8, 12, 3, 7, 0, 4, 10, 1, 13, 11, 6 },
{4, 3, 2, 12, 9, 5, 15, 10, 11, 14, 1, 7, 6, 0, 8, 13 } },
	{ {4, 11, 2, 14, 15, 0, 8, 13, 3, 12, 9, 7, 5, 10, 6, 1},
{13, 0, 11, 7, 4, 9, 1, 10, 14, 3, 5, 12, 2, 15, 8, 6 },
{1, 4, 11, 13, 12, 3, 7, 14, 10, 15, 6, 8, 0, 5, 9, 2 },
{6, 11, 13, 8, 1, 4, 10, 7, 9, 5, 0, 15, 14, 2, 3, 12 } },
	{ {13, 2, 8, 4, 6, 15, 11, 1, 10, 9, 3, 14, 5, 0, 12, 7},
{1, 15, 13, 8, 10, 3, 7, 4, 12, 5, 6, 11, 0, 14, 9, 2 },
{7, 11, 4, 1, 9, 12, 14, 2, 0, 6, 10, 13, 15, 3, 5, 8 },
{2, 1, 14, 7, 4, 10, 8, 13, 15, 12, 9, 0, 3, 5, 6, 11 } }
};

const int P[] = { 15, 6, 19, 20,
	28, 11, 27, 16,
	0, 14, 22, 25,
	4, 17, 30, 9,
	1, 7, 23, 13,
	31, 26, 2, 8,
	18, 12, 29, 5,
	21, 10, 3, 24 };

const int IP_1[] = {
39, 7, 47, 15, 55, 23, 63, 31,
38, 6, 46, 14, 54, 22, 62, 30,
37, 5, 45, 13, 53, 21, 61, 29,
36, 4, 44, 12, 52, 20, 60, 28,
35, 3, 43, 11, 51, 19, 59, 27,
34, 2, 42, 10, 50, 18, 58, 26,
33, 1, 41, 9, 49, 17, 57, 25,
32, 0, 40, 8, 48, 16, 56, 24 };




__constant__ int d_PC_1[56] = { 56, 48, 40, 32, 24, 16, 8, 0,
57, 49, 41, 33, 25, 17, 9, 1,
58, 50, 42, 34, 26, 18, 10, 2,
59, 51, 43, 35, 62, 54, 46, 38,
30, 22, 14, 6, 61, 53, 45, 37,
29, 21, 13, 5, 60, 52, 44, 36,
28, 20, 12, 4, 27, 19, 11, 3 };


__constant__ int d_shifts[] = { 1, 1, 2, 2, 2, 2, 2, 2, 1, 2, 2, 2, 2, 2, 2, 1 };

__constant__ int d_PC_2[] = { 13, 16, 10, 23, 0, 4,
2, 27, 14, 5, 20, 9,
22, 18, 11, 3, 25, 7,
15, 6, 26, 19, 12, 1,
40, 51, 30, 36, 46, 54,
29, 39, 50, 44, 32, 47,
43, 48, 38, 55, 33, 52,
45, 41, 49, 35, 28, 31 };


__constant__ int d_IP[] = { 57, 49, 41, 33, 25, 17, 9, 1,
59, 51, 43, 35, 27, 19, 11, 3,
61, 53, 45, 37, 29, 21, 13, 5,
63, 55, 47, 39, 31, 23, 15, 7,
56, 48, 40, 32, 24, 16,  8, 0,
58, 50, 42, 34, 26, 18, 10, 2,
60, 52, 44, 36, 28, 20, 12, 4,
62, 54, 46, 38, 30, 22, 14, 6 };

__constant__ int d_E[] = { 31, 0, 1, 2, 3, 4,
3, 4, 5, 6, 7, 8,
7, 8, 9, 10, 11, 12,
11, 12, 13, 14, 15, 16,
15, 16, 17, 18, 19, 20,
19, 20, 21, 22, 23, 24,
23, 24, 25, 26, 27, 28,
27, 28, 29, 30, 31, 0 };

__constant__ int d_S[8][4][16] = { { { 14, 4, 13, 1, 2, 15, 11, 8, 3, 10, 6, 12, 5, 9, 0, 7 },
{ 0, 15, 7, 4, 14, 2, 13, 1, 10, 6, 12, 11, 9, 5, 3, 8 },
{ 4, 1, 14, 8, 13, 6, 2, 11, 15, 12, 9, 7, 3, 10, 5, 0 },
{ 15, 12, 8, 2, 4, 9, 1, 7, 5, 11, 3, 14, 10, 0, 6, 13 } },
{ { 15, 1, 8, 14, 6, 11, 3, 4, 9, 7, 2, 13, 12, 0, 5, 10 },
{ 3, 13, 4, 7, 15, 2, 8, 14, 12, 0, 1, 10, 6, 9, 11, 5 },
{ 0, 14, 7, 11, 10, 4, 13, 1, 5, 8, 12, 6, 9, 3, 2, 15 },
{ 13, 8, 10, 1, 3, 15, 4, 2, 11, 6, 7, 12, 0, 5, 14, 9 } },
{ { 10, 0, 9, 14, 6, 3, 15, 5, 1, 13, 12, 7, 11, 4, 2, 8 },
{ 13, 7, 0, 9, 3, 4, 6, 10, 2, 8, 5, 14, 12, 11, 15, 1 },
{ 13, 6, 4, 9, 8, 15, 3, 0, 11, 1, 2, 12, 5, 10, 14, 7 },
{ 1, 10, 13, 0, 6, 9, 8, 7, 4, 15, 14, 3, 11, 5, 2, 12 } },
{ { 7, 13, 14, 3, 0, 6, 9, 10, 1, 2, 8, 5, 11, 12, 4, 15 },
{ 13, 8, 11, 5, 6, 15, 0, 3, 4, 7, 2, 12, 1, 10, 14, 9 },
{ 10, 6, 9, 0, 12, 11, 7, 13, 15, 1, 3, 14, 5, 2, 8, 4 },
{ 3, 15, 0, 6, 10, 1, 13, 8, 9, 4, 5, 11, 12, 7, 2, 14 } },
{ { 2, 12, 4, 1, 7, 10, 11, 6, 8, 5, 3, 15, 13, 0, 14, 9 },
{ 14, 11, 2, 12, 4, 7, 13, 1, 5, 0, 15, 10, 3, 9, 8, 6 },
{ 4, 2, 1, 11, 10, 13, 7, 8, 15, 9, 12, 5, 6, 3, 0, 14 },
{ 11, 8, 12, 7, 1, 14, 2, 13, 6, 15, 0, 9, 10, 4, 5, 3 } },
{ { 12, 1, 10, 15, 9, 2, 6, 8, 0, 13, 3, 4, 14, 7, 5, 11 },
{ 10, 15, 4, 2, 7, 12, 9, 5, 6, 1, 13, 14, 0, 11, 3, 8 },
{ 9, 14, 15, 5, 2, 8, 12, 3, 7, 0, 4, 10, 1, 13, 11, 6 },
{ 4, 3, 2, 12, 9, 5, 15, 10, 11, 14, 1, 7, 6, 0, 8, 13 } },
{ { 4, 11, 2, 14, 15, 0, 8, 13, 3, 12, 9, 7, 5, 10, 6, 1 },
{ 13, 0, 11, 7, 4, 9, 1, 10, 14, 3, 5, 12, 2, 15, 8, 6 },
{ 1, 4, 11, 13, 12, 3, 7, 14, 10, 15, 6, 8, 0, 5, 9, 2 },
{ 6, 11, 13, 8, 1, 4, 10, 7, 9, 5, 0, 15, 14, 2, 3, 12 } },
{ { 13, 2, 8, 4, 6, 15, 11, 1, 10, 9, 3, 14, 5, 0, 12, 7 },
{ 1, 15, 13, 8, 10, 3, 7, 4, 12, 5, 6, 11, 0, 14, 9, 2 },
{ 7, 11, 4, 1, 9, 12, 14, 2, 0, 6, 10, 13, 15, 3, 5, 8 },
{ 2, 1, 14, 7, 4, 10, 8, 13, 15, 12, 9, 0, 3, 5, 6, 11 } }
};

__constant__ int d_P[] = { 15, 6, 19, 20,
28, 11, 27, 16,
0, 14, 22, 25,
4, 17, 30, 9,
1, 7, 23, 13,
31, 26, 2, 8,
18, 12, 29, 5,
21, 10, 3, 24 };

__constant__ int d_IP_1[] = {
	39, 7, 47, 15, 55, 23, 63, 31,
	38, 6, 46, 14, 54, 22, 62, 30,
	37, 5, 45, 13, 53, 21, 61, 29,
	36, 4, 44, 12, 52, 20, 60, 28,
	35, 3, 43, 11, 51, 19, 59, 27,
	34, 2, 42, 10, 50, 18, 58, 26,
	33, 1, 41, 9, 49, 17, 57, 25,
	32, 0, 40, 8, 48, 16, 56, 24 };



void fun()
{
	for(int i = 0; i < 56; i++)
	{
		cout << d_PC_1[i] - 1 << ", ";

		if (!(i % 8))
			cout << "\n";
	}
}

template< typename T, size_t N, size_t M >
void printArray(T(&theArray)[N][M], int char_endl_nbr) {
	for (int x = 0; x < N; x++) 
	{
		for (int y = 0; y < M; y++)
		{
			cout << theArray[x][y];
			if (y == char_endl_nbr)
				cout << endl;
		}
		cout << endl;
	}
}


template< typename T, size_t N, size_t M >
void printArray2(T(&theArray)[N][M], int char_endl_nbr) {
	for (int x = 0; x < N; x++)
	{
		for (int y = 0; y < M; y++)
		{
			cout << theArray[x][y];
			if (y == char_endl_nbr)
				cout << endl;
		}
		cout << endl;
	}
}

__device__ void permutePC(int key_binary[], int key_binary_ret[], int key_binary_size, const int PC[])
{
	for (int i = 0; i < key_binary_size; i++)
		key_binary_ret[i] = key_binary[PC[i]];

}

//C and D should have 28 array memebers
__device__ void createSubkeys(int key[], const int key_size, int C[], int D[], int CD_size, int run_number)
{
	const int size = key_size / 2;
	int tmp_C[28], tmp_D[28];
	for(int i = 0; i < key_size / 2; i++)
	{
		tmp_C[i] = key[i];
		tmp_D[i] = key[i + CD_size];
	}

	for(int i = 0; i < CD_size; i++)
	{
		C[i] = tmp_C[(i + d_shifts[run_number]) % CD_size];
		D[i] = tmp_D[(i + d_shifts[run_number]) % CD_size];
	}

}

__host__ __device__ void decimal2Binary(int decimal_int, int binary_int[], int run_number)
{
	if (decimal_int <= 1) {
		binary_int[run_number] = decimal_int;
		return;
	}

	int remainder = decimal_int % 2;
	decimal2Binary(decimal_int >> 1, binary_int, run_number + 1);
	binary_int[run_number] = remainder;
}

__device__ void reverseTab(int tab[], int tab_length)
{
	for (int i = 0; i < tab_length / 2; i++)
	{
		int tmp = tab[i];
		tab[i] = tab[tab_length - i - 1];
		tab[tab_length - i - 1] = tmp;

	}
}

__device__ void appendKeys(int leftKey[], int rightKey[], int key_size, int key_ret[])
{
	for(int i = 0; i < key_size; i++)
	{
		key_ret[i] = leftKey[i];
		key_ret[i + key_size] = rightKey[i];
	}
}


__device__ void expand(int R[], int tab_ret[], const int E[], int E_size)
{

	for (int i = 0; i < E_size; i++)
		tab_ret[i] = R[E[i]];
}

__device__ void xor(int first_tab[], int second_tab[], int tab_size, int tab_ret[])
{
	for (int i = 0; i < tab_size; i++)
		tab_ret[i] = (int)(!first_tab[i] != !second_tab[i]);

}

//-->
//__device__ long long binary2Decimal(int binary_int[], int tab_length)
//{
//	/*string int_string = "";
//
//	for (int i = 0; i < tab_length; i++)
//		int_string += to_string(binary_int[i]);
//	stringstream ss;
//	ss << int_string;
//	string str = ss.str();
//	unsigned long long value = std::stoull(str, 0, 2);
//	//std::cout << value << std::endl;
//	*/
//	return 0;
//}

__device__ long long binary2Decimal(int binary_int[], int tab_length)
{
	long long dec = 0;

	for (int i = 0; i < tab_length; ++i)
	{
		int bin = binary_int[i];
		if (bin) dec = dec * 2 + 1;
		else dec *= 2;

	}

	return dec;
}

__device__ void f(int R[], int K[], int ret_tab[])
{
	int R_expanded[48];
	expand(R, R_expanded, d_E, 48);
	//DEBUG
//	for (int i = 0; i < 48; i++)
//	{
//		if (!(i % 6))
//			cout << " ";
//		cout << R_expanded[i];
//	}
//	cout << endl << endl << endl;
	
	int xored[48];
	xor (K, R_expanded, 48, xored);
	//DEBUG
//	for(int i = 0; i < 48; i++)
//	{
//		if (!(i % 6))
//			cout << " ";
//		cout << xored[i];
//	}
//	cout << endl << endl << endl;


	for(int i = 0; i < 8; i++)
	{
		int row[4] = {0, 0, 0, 0 }, column[4] = {0, 0, 0, 0};
		row[3] = xored[6 * i + 5];
		row[2] = xored[6 * i];
		column[0] = xored[6 * i + 1];
		column[1] = xored[6 * i + 2];
		column[2] = xored[6 * i + 3];
		column[3] = xored[6 * i + 4];

		int chunk_length = 4;
		int R_chunk[4] = {0, 0, 0, 0};
		decimal2Binary(d_S[i][binary2Decimal(row, 4)][binary2Decimal(column, 4)], R_chunk, 0);
		reverseTab(R_chunk, chunk_length);
		for(int j = 0; j < chunk_length; j++)
		{
			R[4 * i + j] = R_chunk[j];
		}

	}
	//DEBUG
//	for(int i = 0; i < 32; i++)
//	{
//		if (!(i % 4))
//			cout << " ";
//		cout << R[i];
//	}
//	cout << endl << endl << endl;
		
	permutePC(R, ret_tab, 32, d_P);
	//DEBUG
//	for (int i = 0; i < 32; i++)
//	{
//		if (!(i % 4))
//			cout << " ";
//		cout << ret_tab[i];
//	}
//	cout << endl << endl << endl;

}


__device__ void reverse(int L[], int R[], int tab_length, int ret_tab[])
{
	for (int i = 0; i < tab_length; i++)
	{
		ret_tab[i] = R[i];
		ret_tab[i + tab_length] = L[i];
	}
}

__device__ void messageEncode(int message_binary[], int message_size, int K[][48], int msg_ret[])
{
	int L[32], R[32];
	for(int i = 0; i < message_size / 2; i++)
	{
		L[i] = message_binary[i];
		R[i] = message_binary[i + message_size / 2];
	}

	int prev_L[32], prev_R[32];
	for (int i = 0; i < message_size / 2; i++)
	{
		prev_L[i] = L[i];
		prev_R[i] = R[i];
	}

	for(int i = 0; i < 16; i++)
	{	
		for (int j = 0; j < message_size / 2; j++)
			L[j] = prev_R[j];

		int tmp_f[32];
		f(prev_R, K[i], tmp_f);
		//DEBUG
//		for (int i = 0; i < 32; i++)
//		{
//			if (!(i % 4))
//				cout << " ";
//			cout << tmp_f[i];
//		}
//		cout << endl << endl << endl;

		xor(prev_L, tmp_f, 32, R);

		//DEBUG
//		for(int i = 0; i < 32; i++)
//		{
//			if (!(i % 4))
//				cout << " ";
//			cout << R[i];
//		}
//		cout << endl << endl << endl;

		//przepisanie R i L do prev_R i prev_L
		for(int j = 0; j < message_size / 2; j++)
		{
			prev_L[j] = L[j];
			prev_R[j] = R[j];
		}
	}

	int msg[64];
	reverse(L, R, 32, msg);
	//DEBUG
//	for (int i = 0; i < 64; i++)
//	{
//		if (!(i % 8))
//			cout << " ";
//		cout << msg[i];
//	}
	
	permutePC(msg, msg_ret, 64, d_IP_1);
	//DEBUG
//	for(int i = 0; i < 64; i++)
//	{
//		if (!(i % 8))
//			cout << " ";
//		cout << msg_ret[i];
//	}	

}	 





//key_binary_ret should be 64 bit long
__device__ void desEncryption(int message_binary[], int message_size, int key_binary[], int key_size, int msg_ret[])
{
	int des_block_size_bytes = 8;
	int des_block_size_bits = 64;

	//DEBUG
//	printf("\n%s\n", "__device__ desEncryption ");
//	printf("%s\n", "message_binary");
//	for (int i = 0; i < message_size; ++i)
//	{
//		printf("%i", message_binary[i]);
//	}
//		cout << message.size();
//		cout << "\n" << message << "\n";
//if (message.size() * CHAR_BIT != des_block_size_bits)
//		cout << message.size() * CHAR_BIT;
		

	if (message_size % des_block_size_bytes)
	{
		//int tmp_message_binary[message_size + des_block_size_bytes - (message_size % des_block_size_bytes)]
			printf("%s\n", "KICIA");
	//	message_binary.append(des_block_size_bytes - (message.size() % des_block_size_bytes), '0');//mayby another char to append  
	}

	//OLD Verwsion with message as string 
//	if (message.size() % des_block_size_bytes)
//		message.append(des_block_size_bytes - (message.size() % des_block_size_bytes), '0');//mayby another char to append  

	//DEBUG
//		cout << "\n" << message << "\n";
//		cout << message.size();

	int key_binary_ret[56];
	permutePC(key_binary, key_binary_ret, sizeof(key_binary_ret) / sizeof(key_binary_ret[0]), d_PC_1);

	//DEBUG
//	for (int i = 0; i < 56; i++)
//	{
//		if (!(i % 7))
//			cout << "\n";
//		cout << key_binary_ret[i];
//	}

	int subkeys_number = 17;
	int subkey_size = 28;
	int subkeys[17][56];
	int C[28], D[28];

	for (int i = 0; i < 56; i++)
		subkeys[0][i] = key_binary_ret[i];

	for(int i = 0; i < subkeys_number - 1; i++)
	{
		createSubkeys(subkeys[i], sizeof(key_binary_ret) / sizeof(key_binary_ret[0]), C, D, sizeof(C) / sizeof(C[0]), i);
		appendKeys(C, D, subkey_size, subkeys[i + 1]);
		//DEBUG
//		for(int i = 0; i < subkeys_number; i++)
//			for(int j = 0; j < 56; j++)
//				cout << 
//		for (int i = 0; i < 28; i++)
//		{
//			cout << C[i];
//		}
//		cout << endl;
//		for (int i = 0; i < 28; i++)
//		{
//			cout << D[i];
//		}
//		cout << endl;

	}

	//DEBUG
//	printArray(subkeys, 1000);

	int K[16][48];
	for(int i = 0; i < 16; i++)
	{
		permutePC(subkeys[i + 1], K[i], sizeof(K[0]) / sizeof(K[0][0]), d_PC_2);
	}

	//DEBUG
//	printArray2(K, 10000);

	//WARNING!!! message size 
	int message_binary_ret[64];
	permutePC(message_binary, message_binary_ret, message_size, d_IP);

	//DEBUG
//	for(int i = 0; i < message_size; i++)
//		cout << message_binary_ret[i];
	messageEncode(message_binary_ret, message_size, K, msg_ret);

}


void bytes2Bits(vector<BYTE> bytes, int bits[])
{
	for(int i = 0; i < bytes.size(); i++)
	{
		BYTE cur = bytes[i];
		int offset = i * CHAR_BIT;

		for (int bit = 0; bit < CHAR_BIT; bit++, offset++)
		{
			bits[offset] = cur & 1;
			cur >>= 1;  // Move to next bit in array
		}
	}

}

vector<BYTE> hex2Byte(string string_hex)
{
	stringstream converter;
	istringstream istringstream_hex(string_hex);
	vector<BYTE> bytes;

	string word;
	while (istringstream_hex >> word)
	{
		BYTE temp;
		converter << std::hex << word;
		converter >> temp;
		bytes.push_back(temp);
	}

	return bytes;
}

enum DesStringBase
{
	Decimal, 
	Hex,
	Binary//not implemented 
};

const char* hexChar2Bin(char c)
{
	// TODO handle default / error
	switch (toupper(c))
	{
		case '0': return "0000";
		case '1': return "0001";
		case '2': return "0010";
		case '3': return "0011";
		case '4': return "0100";
		case '5': return "0101";
		case '6': return "0110";
		case '7': return "0111";
		case '8': return "1000";
		case '9': return "1001";
		case 'A': return "1010";
		case 'B': return "1011";
		case 'C': return "1100";
		case 'D': return "1101";
		case 'E': return "1110";
		case 'F': return "1111";
	}
}

std::string hex2Bin(const std::string& hex)
{
	// TODO use a loop from <algorithm> or smth
	std::string bin;
	for (unsigned i = 0; i != hex.length(); ++i)
		bin += hexChar2Bin(hex[i]);
	return bin;
}

void str2Int(string& str_int, int ret_int[], int ret_int_size)
{
	for (int i = 0; i < ret_int_size; i++)
		ret_int[i] = (str_int.c_str()[i] - '0');

}

void bin2Hex(string binary)
{
	long int longint = 0;
	for (int i = 0; i < binary.size(); i++)
		longint += (binary[binary.size() - i - 1] - 48) * pow(2, i);
	cout << setbase(16);
	cout << longint;

}

string getHexStringFromBinaryString(string sHex)
{
	string sReturn = "";
	int bit_length = 4;
	const string const bins[] = { "0000", "0001", "0010", "0011", "0100", "0101", "0110", "0111",
		"1000", "1001", "1010", "1011", "1100", "1101", "1110", "1111" };
	for (int i = 0; i < sHex.length() / bit_length; ++i)
	{
		string s = sHex.substr(bit_length * i, bit_length);

		if(s == bins[0])
			 sReturn.append("0");
		if (s == bins[1])
			 sReturn.append("1");
		if (s == bins[2])
			sReturn.append("2");
		if (s == bins[3])
			sReturn.append("3");
		if (s == bins[4])
			sReturn.append("4");
		if (s == bins[5])
			 sReturn.append("5");
		if (s == bins[6])
			 sReturn.append("6");
		if (s == bins[7])
			sReturn.append("7");
		if (s == bins[8])
			sReturn.append("8");
		if (s == bins[9])
			sReturn.append("9");
		if (s == bins[10])
			sReturn.append("A");
		if (s == bins[11])
			sReturn.append("B");
		if (s == bins[12])
			 sReturn.append("C");
		if (s == bins[13])
			 sReturn.append("D");
		if (s == bins[14])
			sReturn.append("E");
		if (s == bins[15])
			 sReturn.append("F");
		}

	return sReturn;

}


string desEncryption(string message2Encrypt, string key, DesStringBase base)
{
	//TODO implement different bases
	string str_message = hex2Bin(message2Encrypt);
	int message_binary_size = 64;
	int message_binary[64];
	str2Int(str_message, message_binary, message_binary_size);
	string str_key = hex2Bin(key);
	int key_binary_size = 64;
	int key_binary[64];
	str2Int(str_key, key_binary, key_binary_size);


	if(base == Decimal)
	{
		//TODO implement decimal to hex
	}

	int msg_ret[64];
	//ANKOMENT MI
//	desEncryption(message_binary, message_binary_size, key_binary, key.size(), msg_ret);
	//ANKOMENT MI
	
	//DEBUG
	//	for (int i = 0; i < 64; i++)
	//	{
	//		if (!(i % 8))
	//			cout << " ";
	//		cout << msg_ret[i];
	//	}

	string binary;
	for (int i = 0; i < 64; i++)
		binary.push_back(std::to_string(msg_ret[i]).c_str()[0]);
	//DEBUG
	//cout << binary;

	return getHexStringFromBinaryString(binary);
}

__host__ __device__ void consecutiveKeyGenerator(unsigned long long &present_key, int next_key_binary[], int next_key_binary_size)
{
	for (int i = 0; i < next_key_binary_size; i++)
		next_key_binary[i] = 0;
	decimal2Binary(present_key, next_key_binary, 0);
	present_key++;
}

__host__ __device__ bool compareArrays(int message[], int cyphertext[])
{
	for (int i = 0; i < 64; i++)
	{
		if (message[i] != cyphertext[i])
			return false;
	}

	return true;
}



__global__ 
void crackDes(int message_binary[], int cyphertext_binary[], int message_binary_size)
{
//	printf("%s\n", "__global__ cracDes");

	int possible_key_binary_size = 56;
	int possible_key_binary[56];
	unsigned long long present_key = 0;
	consecutiveKeyGenerator(present_key, possible_key_binary, possible_key_binary_size);
	//DEBUG
//	for (int i = 0; i < possible_key_binary_size; ++i)
//	{
//		printf("%i", possible_key_binary[i]);
//	}
	
	unsigned long long last_key;
	int msg_ret[64];
	
//	printf("%s\n", "BEFORE desEncryption");

	desEncryption(message_binary, message_binary_size, possible_key_binary, 16, msg_ret);

//	printf("%s\n", "AFTER desEncryption");

	if (compareArrays(msg_ret, cyphertext_binary))
		printf("%s\n", "true");
	else
		printf("%s\n", "false");

	if (compareArrays(msg_ret, cyphertext_binary))
		for (int i = 0; i < possible_key_binary_size; i++)
			printf("%i", possible_key_binary[i]);
	printf("\n");
}

//__host__ 
void IReportU(int message_binary[], int cyphertext_binary[], int message_binary_size)
{
	printf("%s\n", "__global__ cracDes");

	int possible_key_binary_size = 64;
	int possible_key_binary[64];
	unsigned long long present_key = 0;
	consecutiveKeyGenerator(present_key, possible_key_binary, possible_key_binary_size);
	//DEBUG
	int i;
		for (i = 0; i < possible_key_binary_size; ++i)
		{
			printf("%i", possible_key_binary[i]);
		}
	//	printf("%i", i);

	unsigned long long last_key;
	int msg_ret[64];

	printf("%s\n", "BEFORE desEncryption");
//ANKOMENT MI
//	desEncryption(message_binary, message_binary_size, possible_key_binary, 16, msg_ret);
//ANKOMENT MI
	printf("%s\n", "AFTER desEncryption");

	if (compareArrays(msg_ret, cyphertext_binary))
		printf("%s\n", "true");
	else
		printf("%s\n", "false");

	if (compareArrays(msg_ret, cyphertext_binary))
		for (int i = 0; i < possible_key_binary_size; i++)
			printf("%i", possible_key_binary[i]);
	printf("\n");
}


__host__ 
void crackDes(string message, string cyphertext)
{
	string str_message = hex2Bin(message);
	int h_message_binary_size = 64;
	int h_message_binary[64];
	str2Int(str_message, h_message_binary, h_message_binary_size);

	string str_cyphertext = hex2Bin(cyphertext);
	int h_cyphertext_binary_size = 64;
	int h_cyphertext_binary[64];
	str2Int(str_cyphertext, h_cyphertext_binary, h_cyphertext_binary_size);
	
	int* d_message_binary = 0;
	hipMalloc((void**)&d_message_binary, h_message_binary_size * sizeof(int));
	hipMemcpy(d_message_binary, h_message_binary, h_message_binary_size * sizeof(int), hipMemcpyHostToDevice);

	int* d_cyphertext_binary = 0;
	hipMalloc((void**)&d_cyphertext_binary, h_cyphertext_binary_size * sizeof(int));
	hipMemcpy(d_cyphertext_binary, h_cyphertext_binary, h_cyphertext_binary_size * sizeof(int), hipMemcpyHostToDevice);

	//IReportU(h_message_binary, h_cyphertext_binary, h_message_binary_size);
	//TEST1<<<1, 1 >>>();
	crackDes<<<1, 1>>>(d_message_binary, d_cyphertext_binary, h_message_binary_size);

	//DEBUG
	//	for (int i = 0; i < 64; i++)
	//	{
	//		if (!(i % 8))
	//			cout << " ";
	//		cout << msg_ret[i];
	//	}
		
//	string binary;
//	for (int i = 0; i < 64; i++)
//		binary.push_back(std::to_string(msg_ret[i]).c_str()[0]);
	//DEBUG
	//cout << binary;

}

__global__
void desEncryption(int message_binary[], int key_binary[], int message_binary_size, int msg_ret[])
{
	//DEBUG
//	printf("%s\n", "before DEBUG __global__ desEncryption MESSAGE_BINARY");
//	for (int i = 0; i < message_binary_size; ++i)
//	{
//		printf("%i", message_binary[i]);
//	}
//	printf("%s\n", "after DEBUG __global__ desEncryption MESSAGE_BINARY");

	//int msg_ret[64];
//	printf("%s\n", "BEFORE desEncryption");							14 should be here
	desEncryption(message_binary, message_binary_size, key_binary, 16, msg_ret);
//	printf("%s\n", "before DEBUG __global__ desEncryption MSG_RET");
//	for (int i = 0; i < 64; ++i)
//	{
//		printf("%i", 123123123);
//	}
//	printf("%s\n", "after DEBUG __global__ desEncryption MSG_RET");



}
__host__
string desEncryption(string message, string key, char cyphertext[])
{
	string str_message = hex2Bin(message);
	int h_message_binary_size = 64;
	int h_message_binary[64];
	str2Int(str_message, h_message_binary, h_message_binary_size);

	string str_key = hex2Bin(key);
	int h_key_binary_size = 64;
	int h_key_binary[64];
	str2Int(str_key, h_key_binary, h_key_binary_size);

	int* d_message_binary = 0;
	hipMalloc((void**)&d_message_binary, h_message_binary_size * sizeof(int));
	hipMemcpy(d_message_binary, h_message_binary, h_message_binary_size * sizeof(int), hipMemcpyHostToDevice);

	int* d_key_binary = 0;
	hipMalloc((void**)&d_key_binary, h_key_binary_size * sizeof(int));
	hipMemcpy(d_key_binary, h_key_binary, h_key_binary_size * sizeof(int), hipMemcpyHostToDevice);

	int* d_msg_ret;
	hipMalloc((void**)&d_msg_ret, 64 * sizeof(int));

	//DEBUG
//	printf("%s\n", "before DEBUG __host__ desEncryption");
//	for (int i = 0; i < 64; ++i)
//	{
//		printf("%i", h_message_binary[i]);
//	}
//	printf("%s\n", "after DEBUG __host__ desEncryption");

	desEncryption<<<1, 1>>>(d_message_binary, d_key_binary, 64, d_msg_ret);
	hipDeviceSynchronize();

	int* h_msg_ret = (int*)malloc(64 * sizeof(int));
	hipMemcpy(h_msg_ret, d_msg_ret, 64 * sizeof(int), hipMemcpyDeviceToHost);
	//DEBUG
//	printf("\n%s\n", "before DEBUG __host__ desEncryption H_MSG_RET");
//	for (int i = 0; i < 64; ++i)
//	{
//		printf("%i", h_msg_ret[i]);
//	}
//	printf("%s\n", "after DEBUG __host__ desEncryption H_MSG_RET");


	string binary;
	for (int i = 0; i < 64; i++)
		binary.push_back(std::to_string(h_msg_ret[i]).c_str()[0]);
	//DEBUG
	//cout << binary;

	return getHexStringFromBinaryString(binary);
		
}


void initArrays()
{
	hipMemcpyToSymbol(HIP_SYMBOL(d_PC_1), PC_1, PC_1_size * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_shifts), shifts, shifts_size * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_PC_2), PC_2, PC_2_size * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_IP), IP, IP_size * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_E), E, E_size * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_P), P, P_size * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_S), S, S_size_1 * S_size_2 * S_size_3 * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_IP_1), IP_1, IP_1_size * sizeof(int));

}

__global__ void arrayCheck(int size1, int size2, int size3)
{
	for (int i = 0; i < size1; i++)
	{
		for (int j = 0; j < size2; j++)
		{
			for (int k = 0; k < size3; ++k)
			{
				printf("%i,", d_S[i][j][k]);
			}
			printf("\n");
		}
		printf("\n\n");
	}
		
}

long long b2D(int binary_int[], int tab_length)
{
	string int_string = "";

	for (int i = 0; i < tab_length; i++)
		int_string += to_string(binary_int[i]);
	stringstream ss;
	ss << int_string;
	string str = ss.str();
	unsigned long long value = std::stoull(str, 0, 2);
	//std::cout << value << std::endl;
	return value;
}




void reimmplemnt()
{
//	int b1[] = {1, 1, 1, 1, 1, 1, 0, 0, 0,1 ,0 ,0, 1, 1  };
//	long long b2D1 = b2D(b1, 14), B1 = B(b1, 14);
//	printf("%i\n", b2D1);
//	printf("%i\n", B1);


}


bool inRange(int number, int lower_bound, int upper_bound)
{
	return (lower_bound <= number && number >= upper_bound);
}

int main()
{

	size_t size_heap, size_stack;
	hipDeviceSetLimit(hipLimitMallocHeapSize, 20000000 * sizeof(double));
	hipDeviceSetLimit(hipLimitStackSize, 12928);
	hipDeviceGetLimit(&size_heap, hipLimitMallocHeapSize);
	hipDeviceGetLimit(&size_stack, hipLimitStackSize);
	printf("Heap size found to be %d; Stack size found to be %d\n", (int)size_heap, (int)size_stack);
//	reimmplemnt();
	initArrays();

	
	//	arrayCheck << <1, 1 >> > (S_size_1, S_size_2, S_size_3);
//	hipDeviceSynchronize();
	
	
	//string message = "0123456789ABCDEF", key = "0000000000000000";
	string message = "0123456789ABCDEF", key = "00000000000000";
	char cyphertext[64];
	string ct = desEncryption(message, key, cyphertext);
	cout << ct << "\n";
	crackDes(message, ct.c_str());
	hipDeviceSynchronize();


//	for (int i = 0; i < 56; ++i)
//	{
//		if (inRange(PC_1[i], 0, 7))
//			printf("%i,", PC_1[i]);
//		else if (inRange(PC_1[i], 8, 15))
//			printf("%i,", PC_1[i] - 1);
//		else if (inRange(PC_1[i], 16, 23))
//			printf("%i,", PC_1[i] - 2);
//		else if (inRange(PC_1[i], 24, 31))
//			printf("%i,", PC_1[i] - 3);
//		else if (inRange(PC_1[i], 32, 39))
//			printf("%i,", PC_1[i] - 4);
//		else if (inRange(PC_1[i], 40, 47))
//			printf("%i,", PC_1[i] - 5);
//		else if (inRange(PC_1[i], 48, 55))
//			printf("%i,", PC_1[i] - 5);
//		else if (inRange(PC_1[i], 56, 64))
//			printf("%i,", PC_1[i] - 6);
//		else
//			printf("%s", "PIMPU�");
//	}



	return 0;
}



////////////////////////////////////
//
//7 13 14 3 0 6 9 10 1 2 8 5 11 12 4 15
//13 8 11 5 6 15 0 3 4 7 2 12 1 10 14 9
//10 6 9 0 12 11 7 13 15 1 3 14 5 2 8 4
//3 15 0 6 10 1 13 8 9 4 5 11 12 7 2 14
//2 12 4 1 7 10 11 6 8 5 3 15 13 0 14 9
//14 11 2 12 4 7 13 1 5 0 15 10 3 9 8 6
//4 2 1 11 10 13 7 8 15 9 12 5 6 3 0 14
//11 8 12 7 1 14 2 13 6 15 0 9 10 4 5 3
//12 1 10 15 9 2 6 8 0 13 3 4 14 7 5 11
//10 15 4 2 7 12 9 5 6 1 13 14 0 11 3 8
//9 14 15 5 2 8 12 3 7 0 4 10 1 13 11 6
//4 3 2 12 9 5 15 10 11 14 1 7 6 0 8 13
//4 11 2 14 15 0 8 13 3 12 9 7 5 10 6 1
//13 0 11 7 4 9 1 10 14 3 5 12 2 15 8 6
//1 4 11 13 12 3 7 14 10 15 6 8 0 5 9 2
//6 11 13 8 1 4 10 7 9 5 0 15 14 2 3 12
//13 2 8 4 6 15 11 1 10 9 3 14 5 0 12 7
//1 15 13 8 10 3 7 4 12 5 6 11 0 14 9 2
//7 11 4 1 9 12 14 2 0 6 10 13 15 3 5 8
//2 1 14 7 4 10 8 13 15 12 9 0 3 5 6 11




//
//7 13 14 3 0 6 9 10 1 2 8 5 11 12 4 15
//13 8 11 5 6 15 0 3 4 7 2 12 1 10 14 9
//10 6 9 0 12 11 7 13 15 1 3 14 5 2 8 4
//3 15 0 6 10 1 13 8 9 4 5 11 12 7 2 14
//2 12 4 1 7 10 11 6 8 5 3 15 13 0 14 9
//14 11 2 12 4 7 13 1 5 0 15 10 3 9 8 6
//4 2 1 11 10 13 7 8 15 9 12 5 6 3 0 14
//11 8 12 7 1 14 2 13 6 15 0 9 10 4 5 3
//12 1 10 15 9 2 6 8 0 13 3 4 14 7 5 11
//10 15 4 2 7 12 9 5 6 1 13 14 0 11 3 8
//9 14 15 5 2 8 12 3 7 0 4 10 1 13 11 6
//4 3 2 12 9 5 15 10 11 14 1 7 6 0 8 13
//4 11 2 14 15 0 8 13 3 12 9 7 5 10 6 1
//13 0 11 7 4 9 1 10 14 3 5 12 2 15 8 6
//1 4 11 13 12 3 7 14 10 15 6 8 0 5 9 2
//6 11 13 8 1 4 10 7 9 5 0 15 14 2 3 12
//13 2 8 4 6 15 11 1 10 9 3 14 5 0 12 7
//1 15 13 8 10 3 7 4 12 5 6 11 0 14 9 2
//7 11 4 1 9 12 14 2 0 6 10 13 15 3 5 8
//2 1 14 7 4 10 8 13 15 12 9 0 3 5 6 11




/////////////////////////////////////////////////
//Shifts
//11110000110011001010101011110101010101100110011110001111
//11100001100110010101010111111010101011001100111100011110
//11000011001100101010101111110101010110011001111000111101
//00001100110010101010111111110101011001100111100011110101
//00110011001010101011111111000101100110011110001111010101
//11001100101010101111111100000110011001111000111101010101
//00110010101010111111110000111001100111100011110101010101
//11001010101011111111000011000110011110001111010101010110
//00101010101111111100001100111001111000111101010101011001
//01010101011111111000011001100011110001111010101010110011
//01010101111111100001100110011111000111101010101011001100
//01010111111110000110011001011100011110101010101100110011
//01011111111000011001100101010001111010101010110011001111
//01111111100001100110010101010111101010101011001100111100
//11111110000110011001010101011110101010101100110011110001
//11111000011001100101010101111010101010110011001111000111
//11110000110011001010101011110101010101100110011110001111

//K
//000110110000001011101111111111000111000001110010
//011110011010111011011001110110111100100111100101
//010101011111110010001010010000101100111110011001
//011100101010110111010110110110110011010100011101
//011111001110110000000111111010110101001110101000
//011000111010010100111110010100000111101100101111
//111011001000010010110111111101100001100010111100
//111101111000101000111010110000010011101111111011
//111000001101101111101011111011011110011110000001
//101100011111001101000111101110100100011001001111
//001000010101111111010011110111101101001110000110
//011101010111000111110101100101000110011111101001
//100101111100010111010001111110101011101001000001
//010111110100001110110111111100101110011100111010
//101111111001000110001101001111010011111100001010
//110010110011110110001011000011100001011111110101


//MESSAGE AFTER d_IP
//1100110000000000110011001111111111110000101010101111000010101010



//int main()
//{
//	string message = "0123456789ABCDEF", key = "133457799BBCDFF1";
////	int message_binary[] = { 0,0,0,0, 0,0,0,1, 0,0,1,0, 0,0,1,1, 0,1,0,0, 0,1,0,1, 0,1,1,0, 0,1,1,1, 1,0,0,0, 1,0,0,1, 1,0,1,0, 1,0,1,1, 1,1,0,0, 1,1,0,1, 1,1,1,0, 1,1,1,1};
////	int key_binary[] = { 0,0,0,1,0,0,1,1, 0,0,1,1,0,1,0,0, 0,1,0,1,0,1,1,1, 0,1,1,1,1,0,0,1, 1,0,0,1,1,0,1,1, 1,0,1,1,1,1,0,0, 1,1,0,1,1,1,1,1, 1,1,1,1,0,0,0,1 };
//	time_t start = time(nullptr);
//	for(int i = 0; i < 2000; i++)
//		string cyphertext = desEncryption(message, key, DesStringBase::Hex);
//	time_t stop = time(nullptr);
//
//	cout << "\n\n\n" << difftime(stop, start);
//
//	return 0;
//}