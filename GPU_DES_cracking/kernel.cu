#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string>
#include <iostream>
#include <bitset>
#include <sstream>
#include <stdlib.h>
#include <vector>
#include <iomanip>

typedef unsigned char BYTE;

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


////////////////////////////////////////////////////
//int S1[4][16] = { { 14, 4, 13, 1, 2, 15, 11, 8, 3, 10, 6, 12, 5, 9, 0, 7 },
//{ 0, 15, 7, 4, 14, 2, 13, 1, 10, 6, 12, 11, 9, 5, 3, 8 },
//{ 4, 1, 14, 8, 13, 6, 2, 11, 15, 12, 9, 7, 3, 10, 5, 0 },
//{ 15, 12, 8, 2, 4, 9, 1, 7, 5, 11, 3, 14, 10, 0, 6, 13 } };
//
//int S2[4][16] = { { 15, 1, 8, 14, 6, 11, 3, 4, 9, 7, 2, 13, 12, 0, 5, 10 },
//{ 3, 13, 4, 7, 15, 2, 8, 14, 12, 0, 1, 10, 6, 9, 11, 5 },
//{ 0, 14, 7, 11, 10, 4, 13, 1, 5, 8, 12, 6, 9, 3, 2, 15, },
//{ 13, 8, 10, 1, 3, 15, 4, 2, 11, 6, 7, 12, 0, 5, 14, 9 } };
//
//int S3[4][16] = { { 10, 0, 9, 14, 6, 3, 15, 5, 1, 13, 12, 7, 11, 4, 2, 8 },
//{ 13, 7, 0, 9, 3, 4, 6, 10, 2, 8, 5, 14, 12, 11, 15, 1 },
//{ 13, 6, 4, 9, 8, 15, 3, 0, 11, 1, 2, 12, 5, 10, 14, 7 },
//{ 1, 10, 13, 0, 6, 9, 8, 7, 4, 15, 14, 3, 11, 5, 2, 12 } };
//
//int S4[4][16] = { { 7, 13, 14, 3, 0, 6, 9, 10, 1, 2, 8, 5, 11, 12, 4, 15 },
//{ 13, 8, 11, 5, 6, 15, 0, 3, 4, 7, 2, 12, 1, 10, 14, 9 },
//{ 10, 6, 9, 0, 12, 11, 7, 13, 15, 1, 3, 14, 5, 2, 8, 4 },
//{ 3, 15, 0, 6, 10, 1, 13, 8, 9, 4, 5, 11, 12, 7, 2, 14 } };
//
//int S5[4][16] = { { 2, 12, 4, 1, 7, 10, 11, 6, 8, 5, 3, 15, 13, 0, 14, 9 },
//{ 14, 11, 2, 12, 4, 7, 13, 1, 5, 0, 15, 10, 3, 9, 8, 6 },
//{ 4, 2, 1, 11, 10, 13, 7, 8, 15, 9, 12, 5, 6, 3, 0, 14 },
//{ 11, 8, 12, 7, 1, 14, 2, 13, 6, 15, 0, 9, 10, 4, 5, 3 } };
//
//int S6[4][16] = { { 12, 1, 10, 15, 9, 2, 6, 8, 0, 13, 3, 4, 14, 7, 5, 11 },
//{ 10, 15, 4, 2, 7, 12, 9, 5, 6, 1, 13, 14, 0, 11, 3, 8 },
//{ 9, 14, 15, 5, 2, 8, 12, 3, 7, 0, 4, 10, 1, 13, 11, 6 },
//{ 4, 3, 2, 12, 9, 5, 15, 10, 11, 14, 1, 7, 6, 0, 8, 13 } };
//
//int S7[4][16] = { { 4, 11, 2, 14, 15, 0, 8, 13, 3, 12, 9, 7, 5, 10, 6, 1 },
//{ 13, 0, 11, 7, 4, 9, 1, 10, 14, 3, 5, 12, 2, 15, 8, 6 },
//{ 1, 4, 11, 13, 12, 3, 7, 14, 10, 15, 6, 8, 0, 5, 9, 2 },
//{ 6, 11, 13, 8, 1, 4, 10, 7, 9, 5, 0, 15, 14, 2, 3, 12 } };
//
//int S8[4][16] = { { 13, 2, 8, 4, 6, 15, 11, 1, 10, 9, 3, 14, 5, 0, 12, 7 },
//{ 1, 15, 13, 8, 10, 3, 7, 4, 12, 5, 6, 11, 0, 14, 9, 2 },
//{ 7, 11, 4, 1, 9, 12, 14, 2, 0, 6, 10, 13, 15, 3, 5, 8 },
//{ 2, 1, 14, 7, 4, 10, 8, 13, 15, 12, 9, 0, 3, 5, 6, 11 } };


int PC_1[56] = { 56, 48, 40, 32, 24, 16, 8, 0,
57, 49, 41, 33, 25, 17, 9, 1,
58, 50, 42, 34, 26, 18, 10, 2,
59, 51, 43, 35, 62, 54, 46, 38,
30, 22, 14, 6, 61, 53, 45, 37,
29, 21, 13, 5, 60, 52, 44, 36,
28, 20, 12, 4, 27, 19, 11, 3};


int shifts[] = { 1, 1, 2, 2, 2, 2, 2, 2, 1, 2, 2, 2, 2, 2, 2, 1 };

int PC_2[] = { 13, 16, 10, 23, 0, 4,
2, 27, 14, 5, 20, 9,
22, 18, 11, 3, 25, 7,
15, 6, 26, 19, 12, 1,
40, 51, 30, 36, 46, 54,
29, 39, 50, 44, 32, 47,
43, 48, 38, 55, 33, 52,
45, 41, 49, 35, 28, 31 };


int IP[] = { 57, 49, 41, 33, 25, 17, 9, 1,
59, 51, 43, 35, 27, 19, 11, 3,
61, 53, 45, 37, 29, 21, 13, 5,
63, 55, 47, 39, 31, 23, 15, 7,
56, 48, 40, 32, 24, 16,  8, 0,
58, 50, 42, 34, 26, 18, 10, 2,
60, 52, 44, 36, 28, 20, 12, 4,
62, 54, 46, 38, 30, 22, 14, 6 };

int E[] = { 31, 0, 1, 2, 3, 4,
3, 4, 5, 6, 7, 8,
7, 8, 9, 10, 11, 12,
11, 12, 13, 14, 15, 16,
15, 16, 17, 18, 19, 20,
19, 20, 21, 22, 23, 24,
23, 24, 25, 26, 27, 28,
27, 28, 29, 30, 31, 0 };

int S[8][4][16] = { { {14, 4, 13, 1, 2, 15, 11, 8, 3, 10, 6, 12, 5, 9, 0, 7},
{0, 15, 7, 4, 14, 2, 13, 1, 10, 6, 12, 11, 9, 5, 3, 8},
{4, 1, 14, 8, 13, 6, 2, 11, 15, 12, 9, 7, 3, 10, 5, 0 },
{15, 12, 8, 2, 4, 9, 1, 7, 5, 11, 3, 14, 10, 0, 6, 13 } }, 
	{ {15, 1, 8, 14, 6, 11, 3, 4, 9, 7, 2, 13, 12, 0, 5, 10},
{3, 13, 4, 7, 15, 2, 8, 14, 12, 0, 1, 10, 6, 9, 11, 5 },
{0, 14, 7, 11, 10, 4, 13, 1, 5, 8, 12, 6, 9, 3, 2, 15 },
{13, 8, 10, 1, 3, 15, 4, 2, 11, 6, 7, 12, 0, 5, 14, 9 } }, 
	{ {10, 0, 9, 14, 6, 3, 15, 5, 1, 13, 12, 7, 11, 4, 2, 8},
{13, 7, 0, 9, 3, 4, 6, 10, 2, 8, 5, 14, 12, 11, 15, 1 },
{13, 6, 4, 9, 8, 15, 3, 0, 11, 1, 2, 12, 5, 10, 14, 7 },
{1, 10, 13, 0, 6, 9, 8, 7, 4, 15, 14, 3, 11, 5, 2, 12 } }, 
	{ {7, 13, 14, 3, 0, 6, 9, 10, 1, 2, 8, 5, 11, 12, 4, 15},
{13, 8, 11, 5, 6, 15, 0, 3, 4, 7, 2, 12, 1, 10, 14, 9},
{10, 6, 9, 0, 12, 11, 7, 13, 15, 1, 3, 14, 5, 2, 8, 4},
{3, 15, 0, 6, 10, 1, 13, 8, 9, 4, 5, 11, 12, 7, 2, 14} }, 
	{ {2, 12, 4, 1, 7, 10, 11, 6, 8, 5, 3, 15, 13, 0, 14, 9},
{14, 11, 2, 12, 4, 7, 13, 1, 5, 0, 15, 10, 3, 9, 8, 6},
{4, 2, 1, 11, 10, 13, 7, 8, 15, 9, 12, 5, 6, 3, 0, 14},
{11, 8, 12, 7, 1, 14, 2, 13, 6, 15, 0, 9, 10, 4, 5, 3} }, 
	{ {12, 1, 10, 15, 9, 2, 6, 8, 0, 13, 3, 4, 14, 7, 5, 11},
{10, 15, 4, 2, 7, 12, 9, 5, 6, 1, 13, 14, 0, 11, 3, 8 },
{9, 14, 15, 5, 2, 8, 12, 3, 7, 0, 4, 10, 1, 13, 11, 6 },
{4, 3, 2, 12, 9, 5, 15, 10, 11, 14, 1, 7, 6, 0, 8, 13 } },
	{ {4, 11, 2, 14, 15, 0, 8, 13, 3, 12, 9, 7, 5, 10, 6, 1},
{13, 0, 11, 7, 4, 9, 1, 10, 14, 3, 5, 12, 2, 15, 8, 6 },
{1, 4, 11, 13, 12, 3, 7, 14, 10, 15, 6, 8, 0, 5, 9, 2 },
{6, 11, 13, 8, 1, 4, 10, 7, 9, 5, 0, 15, 14, 2, 3, 12 } },
	{ {13, 2, 8, 4, 6, 15, 11, 1, 10, 9, 3, 14, 5, 0, 12, 7},
{1, 15, 13, 8, 10, 3, 7, 4, 12, 5, 6, 11, 0, 14, 9, 2 },
{7, 11, 4, 1, 9, 12, 14, 2, 0, 6, 10, 13, 15, 3, 5, 8 },
{2, 1, 14, 7, 4, 10, 8, 13, 15, 12, 9, 0, 3, 5, 6, 11 } }
};

int P[] ={15, 6, 19, 20,
	28, 11, 27, 16,
	0, 14, 22, 25,
	4, 17, 30, 9,
	1, 7, 23, 13,
	31, 26, 2, 8,
	18, 12, 29, 5,
	21, 10, 3, 24};

int IP_1[] = {
39, 7, 47, 15, 55, 23, 63, 31,
38, 6, 46, 14, 54, 22, 62, 30,
37, 5, 45, 13, 53, 21, 61, 29,
36, 4, 44, 12, 52, 20, 60, 28,
35, 3, 43, 11, 51, 19, 59, 27,
34, 2, 42, 10, 50, 18, 58, 26,
33, 1, 41, 9, 49, 17, 57, 25,
32, 0, 40, 8, 48, 16, 56, 24 };

void fun()
{
	for(int i = 0; i < 56; i++)
	{
		cout << PC_1[i] - 1 << ", ";

		if (!(i % 8))
			cout << "\n";
	}
}

template< typename T, size_t N, size_t M >
void printArray(T(&theArray)[N][M], int char_endl_nbr) {
	for (int x = 0; x < N; x++) 
	{
		for (int y = 0; y < M; y++)
		{
			cout << theArray[x][y];
			if (y == char_endl_nbr)
				cout << endl;
		}
		cout << endl;
	}
}


template< typename T, size_t N, size_t M >
void printArray2(T(&theArray)[N][M], int char_endl_nbr) {
	for (int x = 0; x < N; x++)
	{
		for (int y = 0; y < M; y++)
		{
			cout << theArray[x][y];
			if (y == char_endl_nbr)
				cout << endl;
		}
		cout << endl;
	}
}

void permutePC(int key_binary[], int key_binary_ret[], int key_binary_size, int PC[])
{
	for (int i = 0; i < key_binary_size; i++)
		key_binary_ret[i] = key_binary[PC[i]];

}

//C and D should have 28 array memebers
void createSubkeys(int key[], const int key_size, int C[], int D[], int CD_size, int run_number)
{
	const int size = key_size / 2;
	int tmp_C[28], tmp_D[28];
	for(int i = 0; i < key_size / 2; i++)
	{
		tmp_C[i] = key[i];
		tmp_D[i] = key[i + CD_size];
	}

	for(int i = 0; i < CD_size; i++)
	{
		C[i] = tmp_C[(i + shifts[run_number]) % CD_size];
		D[i] = tmp_D[(i + shifts[run_number]) % CD_size];
	}

}

void decimal2Binary(int decimal_int, int binary_int[], int run_number)
{
	if (decimal_int <= 1) {
		binary_int[run_number] = decimal_int;
		return;
	}

	int remainder = decimal_int % 2;
	decimal2Binary(decimal_int >> 1, binary_int, run_number + 1);
	binary_int[run_number] = remainder;
}

void reverseTab(int tab[], int tab_length)
{
	for (int i = 0; i < tab_length / 2; i++)
	{
		int tmp = tab[i];
		tab[i] = tab[tab_length - i - 1];
		tab[tab_length - i - 1] = tmp;

	}
}

void appendKeys(int leftKey[], int rightKey[], int key_size, int key_ret[])
{
	for(int i = 0; i < key_size; i++)
	{
		key_ret[i] = leftKey[i];
		key_ret[i + key_size] = rightKey[i];
	}
}


void expand(int R[], int tab_ret[], int E[], int E_size)
{

	for (int i = 0; i < E_size; i++)
		tab_ret[i] = R[E[i]];
}

void xor(int first_tab[], int second_tab[], int tab_size, int tab_ret[])
{
	for (int i = 0; i < tab_size; i++)
		tab_ret[i] = (int)(!first_tab[i] != !second_tab[i]);

}

//-->
long long binary2Decimal(int binary_int[], int tab_length)
{
	string int_string = "";

	for (int i = 0; i < tab_length; i++)
		int_string += to_string(binary_int[i]);
	stringstream ss;
	ss << int_string;
	string str = ss.str();
	unsigned long long value = std::stoull(str, 0, 2);
	//std::cout << value << std::endl;
	return value;
}

void f(int R[], int K[], int ret_tab[])
{
	int R_expanded[48];
	expand(R, R_expanded, E, 48);
	//DEBUG
//	for (int i = 0; i < 48; i++)
//	{
//		if (!(i % 6))
//			cout << " ";
//		cout << R_expanded[i];
//	}
//	cout << endl << endl << endl;
	
	int xored[48];
	xor (K, R_expanded, 48, xored);
	//DEBUG
//	for(int i = 0; i < 48; i++)
//	{
//		if (!(i % 6))
//			cout << " ";
//		cout << xored[i];
//	}
//	cout << endl << endl << endl;


	for(int i = 0; i < 8; i++)
	{
		int row[4] = {0, 0, 0, 0 }, column[4] = {0, 0, 0, 0};
		row[3] = xored[6 * i + 5];
		row[2] = xored[6 * i];
		column[0] = xored[6 * i + 1];
		column[1] = xored[6 * i + 2];
		column[2] = xored[6 * i + 3];
		column[3] = xored[6 * i + 4];

		int chunk_length = 4;
		int R_chunk[4] = {0, 0, 0, 0};
		decimal2Binary(S[i][binary2Decimal(row, 4)][binary2Decimal(column, 4)], R_chunk, 0);
		reverseTab(R_chunk, chunk_length);
		for(int j = 0; j < chunk_length; j++)
		{
			R[4 * i + j] = R_chunk[j];
		}

	}
	//DEBUG
//	for(int i = 0; i < 32; i++)
//	{
//		if (!(i % 4))
//			cout << " ";
//		cout << R[i];
//	}
//	cout << endl << endl << endl;
		
	permutePC(R, ret_tab, 32, P);
	//DEBUG
//	for (int i = 0; i < 32; i++)
//	{
//		if (!(i % 4))
//			cout << " ";
//		cout << ret_tab[i];
//	}
//	cout << endl << endl << endl;

}


void reverse(int L[], int R[], int tab_length, int ret_tab[])
{
	for (int i = 0; i < tab_length; i++)
	{
		ret_tab[i] = R[i];
		ret_tab[i + tab_length] = L[i];
	}
}

void messageEncode(int message_binary[], int message_size, int K[][48], int msg_ret[])
{
	int L[32], R[32];
	for(int i = 0; i < message_size / 2; i++)
	{
		L[i] = message_binary[i];
		R[i] = message_binary[i + message_size / 2];
	}

	int prev_L[32], prev_R[32];
	for (int i = 0; i < message_size / 2; i++)
	{
		prev_L[i] = L[i];
		prev_R[i] = R[i];
	}

	for(int i = 0; i < 16; i++)
	{	
		for (int j = 0; j < message_size / 2; j++)
			L[j] = prev_R[j];

		int tmp_f[32];
		f(prev_R, K[i], tmp_f);
		//DEBUG
//		for (int i = 0; i < 32; i++)
//		{
//			if (!(i % 4))
//				cout << " ";
//			cout << tmp_f[i];
//		}
//		cout << endl << endl << endl;

		xor(prev_L, tmp_f, 32, R);

		//DEBUG
//		for(int i = 0; i < 32; i++)
//		{
//			if (!(i % 4))
//				cout << " ";
//			cout << R[i];
//		}
//		cout << endl << endl << endl;

		//przepisanie R i L do prev_R i prev_L
		for(int j = 0; j < message_size / 2; j++)
		{
			prev_L[j] = L[j];
			prev_R[j] = R[j];
		}
	}

	int msg[64];
	reverse(L, R, 32, msg);
	//DEBUG
//	for (int i = 0; i < 64; i++)
//	{
//		if (!(i % 8))
//			cout << " ";
//		cout << msg[i];
//	}
	
	permutePC(msg, msg_ret, 64, IP_1);
	//DEBUG
//	for(int i = 0; i < 64; i++)
//	{
//		if (!(i % 8))
//			cout << " ";
//		cout << msg_ret[i];
//	}	

}	 





//key_binary_ret should be 64 bit long
void desEncyption(int message_binary[], int message_size,int key_binary[], int key_size, int msg_ret[])
{
	int des_block_size_bytes = 8;
	int des_block_size_bits = 64;

//	cout << "omg";
	//DEBUG
//		cout << message.size();
//		cout << "\n" << message << "\n";
//if (message.size() * CHAR_BIT != des_block_size_bits)
//		cout << message.size() * CHAR_BIT;
		

	if (message_size % des_block_size_bytes)
	{
		//int tmp_message_binary[message_size + des_block_size_bytes - (message_size % des_block_size_bytes)]
			cout << "KICIA";
	//	message_binary.append(des_block_size_bytes - (message.size() % des_block_size_bytes), '0');//mayby another char to append  
	}

	//OLD Verwsion with message as string 
//	if (message.size() % des_block_size_bytes)
//		message.append(des_block_size_bytes - (message.size() % des_block_size_bytes), '0');//mayby another char to append  

	//DEBUG
//		cout << "\n" << message << "\n";
//		cout << message.size();

	int key_binary_ret[56];
	permutePC(key_binary, key_binary_ret, sizeof(key_binary_ret) / sizeof(key_binary_ret[0]), PC_1);

	//DEBUG
//	for (int i = 0; i < 56; i++)
//	{
//		if (!(i % 7))
//			cout << "\n";
//		cout << key_binary_ret[i];
//	}

	int subkeys_number = 17;
	int subkey_size = 28;
	int subkeys[17][56];
	int C[28], D[28];

	for (int i = 0; i < 56; i++)
		subkeys[0][i] = key_binary_ret[i];

	for(int i = 0; i < subkeys_number - 1; i++)
	{
		createSubkeys(subkeys[i], sizeof(key_binary_ret) / sizeof(key_binary_ret[0]), C, D, sizeof(C) / sizeof(C[0]), i);
		appendKeys(C, D, subkey_size, subkeys[i + 1]);
		//DEBUG
//		for(int i = 0; i < subkeys_number; i++)
//			for(int j = 0; j < 56; j++)
//				cout << 
//		for (int i = 0; i < 28; i++)
//		{
//			cout << C[i];
//		}
//		cout << endl;
//		for (int i = 0; i < 28; i++)
//		{
//			cout << D[i];
//		}
//		cout << endl;

	}

	//DEBUG
//	printArray(subkeys, 1000);

	int K[16][48];
	for(int i = 0; i < 16; i++)
	{
		permutePC(subkeys[i + 1], K[i], sizeof(K[0]) / sizeof(K[0][0]), PC_2);
	}

	//DEBUG
//	printArray2(K, 10000);

	//WARNING!!! message size 
	int message_binary_ret[64];
	permutePC(message_binary, message_binary_ret, message_size, IP);

	//DEBUG
//	for(int i = 0; i < message_size; i++)
//		cout << message_binary_ret[i];
	messageEncode(message_binary_ret, message_size, K, msg_ret);

}


void bytes2Bits(vector<BYTE> bytes, int bits[])
{
	for(int i = 0; i < bytes.size(); i++)
	{
		BYTE cur = bytes[i];
		int offset = i * CHAR_BIT;

		for (int bit = 0; bit < CHAR_BIT; bit++, offset++)
		{
			bits[offset] = cur & 1;
			cur >>= 1;  // Move to next bit in array
		}
	}

}

vector<BYTE> hex2Byte(string string_hex)
{
	stringstream converter;
	istringstream istringstream_hex(string_hex);
	vector<BYTE> bytes;

	string word;
	while (istringstream_hex >> word)
	{
		BYTE temp;
		converter << std::hex << word;
		converter >> temp;
		bytes.push_back(temp);
	}

	return bytes;
}

enum DesStringBase
{
	Decimal, 
	Hex,
	Binary//not implemented 
};

const char* hexChar2Bin(char c)
{
	// TODO handle default / error
	switch (toupper(c))
	{
		case '0': return "0000";
		case '1': return "0001";
		case '2': return "0010";
		case '3': return "0011";
		case '4': return "0100";
		case '5': return "0101";
		case '6': return "0110";
		case '7': return "0111";
		case '8': return "1000";
		case '9': return "1001";
		case 'A': return "1010";
		case 'B': return "1011";
		case 'C': return "1100";
		case 'D': return "1101";
		case 'E': return "1110";
		case 'F': return "1111";
	}
}

std::string hex2Bin(const std::string& hex)
{
	// TODO use a loop from <algorithm> or smth
	std::string bin;
	for (unsigned i = 0; i != hex.length(); ++i)
		bin += hexChar2Bin(hex[i]);
	return bin;
}

vector<int> str2Int(string& str_int)
{
	vector<int> int_vector;
	for (int i = 0; i < str_int.size(); i++)
		int_vector.push_back(str_int.c_str()[i] - '0');

	return int_vector;
}

void bin2Hex(string binary)
{
	long int longint = 0;
	for (int i = 0; i < binary.size(); i++)
		longint += (binary[binary.size() - i - 1] - 48) * pow(2, i);
	cout << setbase(16);
	cout << longint;

}

string getHexStringFromBinaryString(string sHex)
{
	string sReturn = "";
	int bit_length = 4;
	const string const bins[] = { "0000", "0001", "0010", "0011", "0100", "0101", "0110", "0111",
		"1000", "1001", "1010", "1011", "1100", "1101", "1110", "1111" };
	for (int i = 0; i < sHex.length() / bit_length; ++i)
	{
		string s = sHex.substr(bit_length * i, bit_length);

		if(s == bins[0])
			 sReturn.append("0");
		if (s == bins[1])
			 sReturn.append("1");
		if (s == bins[2])
			sReturn.append("2");
		if (s == bins[3])
			sReturn.append("3");
		if (s == bins[4])
			sReturn.append("4");
		if (s == bins[5])
			 sReturn.append("5");
		if (s == bins[6])
			 sReturn.append("6");
		if (s == bins[7])
			sReturn.append("7");
		if (s == bins[8])
			sReturn.append("8");
		if (s == bins[9])
			sReturn.append("9");
		if (s == bins[10])
			sReturn.append("A");
		if (s == bins[11])
			sReturn.append("B");
		if (s == bins[12])
			 sReturn.append("C");
		if (s == bins[13])
			 sReturn.append("D");
		if (s == bins[14])
			sReturn.append("E");
		if (s == bins[15])
			 sReturn.append("F");
		}

	return sReturn;

}


string desEncyption(string message2Encrypt, string key, DesStringBase base)
{
	//TODO implement different bases
	string str_message = hex2Bin(message2Encrypt);
	vector<int> message_binary = str2Int(str_message);
	string str_key = hex2Bin(key);
	vector<int> key_binary = str2Int(str_key);

	if(base == Decimal)
	{
		//TODO implement decimal to hex
	}

	int msg_ret[64];
	desEncyption(&message_binary[0], message_binary.size(), &key_binary[0], key.size(), msg_ret);
	//DEBUG
	//	for (int i = 0; i < 64; i++)
	//	{
	//		if (!(i % 8))
	//			cout << " ";
	//		cout << msg_ret[i];
	//	}

	string binary;
	for (int i = 0; i < 64; i++)
		binary.push_back(std::to_string(msg_ret[i]).c_str()[0]);
	//DEBUG
	//cout << binary;

	return getHexStringFromBinaryString(binary);
}


//Shifts
//11110000110011001010101011110101010101100110011110001111
//11100001100110010101010111111010101011001100111100011110
//11000011001100101010101111110101010110011001111000111101
//00001100110010101010111111110101011001100111100011110101
//00110011001010101011111111000101100110011110001111010101
//11001100101010101111111100000110011001111000111101010101
//00110010101010111111110000111001100111100011110101010101
//11001010101011111111000011000110011110001111010101010110
//00101010101111111100001100111001111000111101010101011001
//01010101011111111000011001100011110001111010101010110011
//01010101111111100001100110011111000111101010101011001100
//01010111111110000110011001011100011110101010101100110011
//01011111111000011001100101010001111010101010110011001111
//01111111100001100110010101010111101010101011001100111100
//11111110000110011001010101011110101010101100110011110001
//11111000011001100101010101111010101010110011001111000111
//11110000110011001010101011110101010101100110011110001111

//K
//000110110000001011101111111111000111000001110010
//011110011010111011011001110110111100100111100101
//010101011111110010001010010000101100111110011001
//011100101010110111010110110110110011010100011101
//011111001110110000000111111010110101001110101000
//011000111010010100111110010100000111101100101111
//111011001000010010110111111101100001100010111100
//111101111000101000111010110000010011101111111011
//111000001101101111101011111011011110011110000001
//101100011111001101000111101110100100011001001111
//001000010101111111010011110111101101001110000110
//011101010111000111110101100101000110011111101001
//100101111100010111010001111110101011101001000001
//010111110100001110110111111100101110011100111010
//101111111001000110001101001111010011111100001010
//110010110011110110001011000011100001011111110101


//MESSAGE AFTER IP
//1100110000000000110011001111111111110000101010101111000010101010



//int main()
//{
//	string message = "0123456789ABCDEF", key = "133457799BBCDFF1";
////	int message_binary[] = { 0,0,0,0, 0,0,0,1, 0,0,1,0, 0,0,1,1, 0,1,0,0, 0,1,0,1, 0,1,1,0, 0,1,1,1, 1,0,0,0, 1,0,0,1, 1,0,1,0, 1,0,1,1, 1,1,0,0, 1,1,0,1, 1,1,1,0, 1,1,1,1};
////	int key_binary[] = { 0,0,0,1,0,0,1,1, 0,0,1,1,0,1,0,0, 0,1,0,1,0,1,1,1, 0,1,1,1,1,0,0,1, 1,0,0,1,1,0,1,1, 1,0,1,1,1,1,0,0, 1,1,0,1,1,1,1,1, 1,1,1,1,0,0,0,1 };
//	time_t start = time(nullptr);
//	for(int i = 0; i < 2000; i++)
//		string cypherText = desEncyption(message, key, DesStringBase::Hex);
//	time_t stop = time(nullptr);
//
//	cout << "\n\n\n" << difftime(stop, start);
//
//	return 0;
//}

vector<int> consecutiveKeyGenerator()
{
	vector<int> key;
	for (int i = 0; i < 63; i++)
		key.push_back(0);
	key.push_back(1);
	return key;
}

bool compareArrays(int message[], vector<int> cypherText)
{
	for (int i = 0; i < 64; i++)
	{
		if (message[i] != cypherText[i])
			return false;
	}

	return true;
}

//__global__ 
__host__ void crackDes(string message, string cyphertext)
{
	string str_message = hex2Bin(message);
	vector<int> message_binary = str2Int(str_message);
	vector<int> possible_key_binary = consecutiveKeyGenerator();
	
	string str_cyphertext = hex2Bin(cyphertext);
	vector<int> cyphertext_binary = str2Int(str_cyphertext);

	int msg_ret[64];
	desEncyption(&message_binary[0], message_binary.size(), &possible_key_binary[0], 16, msg_ret);

	if (compareArrays(msg_ret, cyphertext_binary))
		for (int i = 0; i < 64; i++)
			cout << possible_key_binary[i];
	cout << "\n";
	//DEBUG
	//	for (int i = 0; i < 64; i++)
	//	{
	//		if (!(i % 8))
	//			cout << " ";
	//		cout << msg_ret[i];
	//	}

//	string binary;
//	for (int i = 0; i < 64; i++)
//		binary.push_back(std::to_string(msg_ret[i]).c_str()[0]);
	//DEBUG
	//cout << binary;

}

int main()
{
	string message = "0123456789ABCDEE", key = "0000000000000000";
	string cypherText = desEncyption(message, key, DesStringBase::Hex);
	//cout << cypherText << "\n";
	crackDes(message, cypherText);



//	const int arraySize = 5;
//	const int a[arraySize] = { 1, 2, 3, 4, 5 };
//	const int b[arraySize] = { 10, 20, 30, 40, 50 };
//	int c[arraySize] = { 0 };
//
//	// Add vectors in parallel.
//	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "addWithCuda failed!");
//		return 1;
//	}
//
//
//	// hipDeviceReset must be called before exiting in order for profiling and
//	// tracing tools such as Nsight and Visual Profiler to show complete traces.
//	cudaStatus = hipDeviceReset();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipDeviceReset failed!");
//		return 1;
//	}

	return 0;
}
//
//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//	int *dev_a = 0;
//	int *dev_b = 0;
//	int *dev_c = 0;
//	hipError_t cudaStatus;
//
//	// Choose which GPU to run on, change this on a multi-GPU system.
//	cudaStatus = hipSetDevice(0);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//		goto Error;
//	}
//
//	// Allocate GPU buffers for three vectors (two input, one output)    .
//	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//
//	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//
//	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//
//	// Copy input vectors from host memory to GPU buffers.
//	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto Error;
//	}
//
//	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto Error;
//	}
//
//	// Launch a kernel on the GPU with one thread for each element.
//	crackDes << <1, size >> >(dev_c, dev_a, dev_b);
//
//	// Check for any errors launching the kernel
//	cudaStatus = hipGetLastError();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "crackDes launch failed: %s\n", hipGetErrorString(cudaStatus));
//		goto Error;
//	}
//
//	// hipDeviceSynchronize waits for the kernel to finish, and returns
//	// any errors encountered during the launch.
//	cudaStatus = hipDeviceSynchronize();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching crackDes!\n", cudaStatus);
//		goto Error;
//	}
//
//	// Copy output vector from GPU buffer to host memory.
//	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto Error;
//	}
//
//Error:
//	hipFree(dev_c);
//	hipFree(dev_a);
//	hipFree(dev_b);
//
//	return cudaStatus;
//}



////////////////////////////////////
//
//7 13 14 3 0 6 9 10 1 2 8 5 11 12 4 15
//13 8 11 5 6 15 0 3 4 7 2 12 1 10 14 9
//10 6 9 0 12 11 7 13 15 1 3 14 5 2 8 4
//3 15 0 6 10 1 13 8 9 4 5 11 12 7 2 14
//2 12 4 1 7 10 11 6 8 5 3 15 13 0 14 9
//14 11 2 12 4 7 13 1 5 0 15 10 3 9 8 6
//4 2 1 11 10 13 7 8 15 9 12 5 6 3 0 14
//11 8 12 7 1 14 2 13 6 15 0 9 10 4 5 3
//12 1 10 15 9 2 6 8 0 13 3 4 14 7 5 11
//10 15 4 2 7 12 9 5 6 1 13 14 0 11 3 8
//9 14 15 5 2 8 12 3 7 0 4 10 1 13 11 6
//4 3 2 12 9 5 15 10 11 14 1 7 6 0 8 13
//4 11 2 14 15 0 8 13 3 12 9 7 5 10 6 1
//13 0 11 7 4 9 1 10 14 3 5 12 2 15 8 6
//1 4 11 13 12 3 7 14 10 15 6 8 0 5 9 2
//6 11 13 8 1 4 10 7 9 5 0 15 14 2 3 12
//13 2 8 4 6 15 11 1 10 9 3 14 5 0 12 7
//1 15 13 8 10 3 7 4 12 5 6 11 0 14 9 2
//7 11 4 1 9 12 14 2 0 6 10 13 15 3 5 8
//2 1 14 7 4 10 8 13 15 12 9 0 3 5 6 11




//
//7 13 14 3 0 6 9 10 1 2 8 5 11 12 4 15
//13 8 11 5 6 15 0 3 4 7 2 12 1 10 14 9
//10 6 9 0 12 11 7 13 15 1 3 14 5 2 8 4
//3 15 0 6 10 1 13 8 9 4 5 11 12 7 2 14
//2 12 4 1 7 10 11 6 8 5 3 15 13 0 14 9
//14 11 2 12 4 7 13 1 5 0 15 10 3 9 8 6
//4 2 1 11 10 13 7 8 15 9 12 5 6 3 0 14
//11 8 12 7 1 14 2 13 6 15 0 9 10 4 5 3
//12 1 10 15 9 2 6 8 0 13 3 4 14 7 5 11
//10 15 4 2 7 12 9 5 6 1 13 14 0 11 3 8
//9 14 15 5 2 8 12 3 7 0 4 10 1 13 11 6
//4 3 2 12 9 5 15 10 11 14 1 7 6 0 8 13
//4 11 2 14 15 0 8 13 3 12 9 7 5 10 6 1
//13 0 11 7 4 9 1 10 14 3 5 12 2 15 8 6
//1 4 11 13 12 3 7 14 10 15 6 8 0 5 9 2
//6 11 13 8 1 4 10 7 9 5 0 15 14 2 3 12
//13 2 8 4 6 15 11 1 10 9 3 14 5 0 12 7
//1 15 13 8 10 3 7 4 12 5 6 11 0 14 9 2
//7 11 4 1 9 12 14 2 0 6 10 13 15 3 5 8
//2 1 14 7 4 10 8 13 15 12 9 0 3 5 6 11